#include "hip/hip_runtime.h"
#include <jni.h>
#include <string>
#include <cstddef>
#include <stdexcept>
#include <algorithm>
#include <vector>
#include <limits>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <cstdlib> // for Malloc function

/* --------------------------------------------------------------------------------------------------------
NOTES: 

This gpufit code is consolidated from an open-source code. Please see https://gpufit.readthedocs.io/en/latest/ for more information. 
All codes (headers and implementations), including CUDA codes, are deliberately placed together for ease of compilation 
using nvcc compiler and for ease of distribution. We added ACF and bleach correction fitting functions. Furthermore, 
we have also placed the CUDA ACF calculations codes here. 

Some implementation details:
1. void calcacf3 kernel function calculates the block transformation values of the intensity.
2. void calcacf2a kernel function calculates the arrays according to different time bins in different parts of the correlation function.
3. void calcacf2b kernel function calculates the value of the auto or cross-correlation at every lag time. This function also performs the G1 analysis in N and B calculation.
4. void calc_data_bleach_correction kernel function is an averaging step in temporal dimension for every ave number of points, prior to performing bleach correction fitting.
5. void calc_binning kernel function performs binning of spatial data.
6. void bleachcorrection kernel function performs polynomial bleach correction given polynomial order and coefficients. It is done prior to calcacf3, calcacf2a and calcacf2b.
7. Kindly also take a look at the CPU functions for a detailed description of the variables.
8. argument float* data is the intensity array input on which the N and B or the autocorrelation or the cross-correlation has to be calculates.
9. argument double* data1 is the output array where the values of auto and cross-correlation are calculated.
--------------------------------------------------------------------------------------------------------- */

// declare as a flag during compilation, ie. -D USE_CUBLAS
// Without USE_CUBLAS, we are using solve_equation_systems_gj(), i.e. cuda_gaussjordan.cu
// #define USE_CUBLAS 

// from gpufit.h
#ifdef __linux__
    #define VISIBLE __attribute__((visibility("default")))
#else
    #define VISIBLE
#endif

/* -------------------------------------------------------------------------------------------------------
* from definitions.h START
------------------------------------------------------------------------------------------------------- */
// Precision
#ifdef GPUFIT_DOUBLE
    #define REAL double
#else
    #define REAL float
#endif // GPUFIT_DOUBLE

#ifdef USE_CUBLAS
    #include "hipblas.h"

    #ifdef GPUFIT_DOUBLE
        #define DECOMPOSE_LUP hipblasDgetrfBatched
        #define SOLVE_LUP hipblasDgetrsBatched
    #else
        #define DECOMPOSE_LUP hipblasSgetrfBatched
        #define SOLVE_LUP hipblasSgetrsBatched
    #endif

    #define SOLVE_EQUATION_SYSTEMS() solve_equation_systems_lup()
#else
    #define hipblasHandle_t int
    #define SOLVE_EQUATION_SYSTEMS() solve_equation_systems_gj()
#endif

// Status
#define CUDA_CHECK_STATUS( cuda_function_call ) \
    if (hipError_t const status = cuda_function_call) \
    { \
        throw std::runtime_error( hipGetErrorString( status ) ) ; \
    }
/* -------------------------------------------------------------------------------------------------------
* from definitions.h END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from gpufitImFCS_GpufitImFCS.h START
------------------------------------------------------------------------------------------------------- */
#ifdef __cplusplus
extern "C" {
#endif
/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    fit
 * Signature: (IILjava/nio/FloatBuffer;Ljava/nio/FloatBuffer;ILjava/nio/FloatBuffer;FIILjava/nio/IntBuffer;IILjava/nio/FloatBuffer;Ljava/nio/FloatBuffer;Ljava/nio/IntBuffer;Ljava/nio/FloatBuffer;Ljava/nio/IntBuffer;)I
 */
JNIEXPORT jint JNICALL Java_gpufitImFCS_GpufitImFCS_fit
  (JNIEnv *, jclass, jint, jint, jobject, jobject, jint, jobject, jfloat, jint, jint, jobject, jint, jint, jobject, jobject, jobject, jobject, jobject);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    getLastError
 * Signature: ()Ljava/lang/String;
 */
JNIEXPORT jstring JNICALL Java_gpufitImFCS_GpufitImFCS_getLastError
  (JNIEnv *, jclass);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    isCudaAvailableInt
 * Signature: ()Z
 */
JNIEXPORT jboolean JNICALL Java_gpufitImFCS_GpufitImFCS_isCudaAvailableInt
  (JNIEnv *, jclass);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    getCudaVersionAsArray
 * Signature: ()[I
 */
JNIEXPORT jintArray JNICALL Java_gpufitImFCS_GpufitImFCS_getCudaVersionAsArray
  (JNIEnv *, jclass);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    resetGPU
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_gpufitImFCS_GpufitImFCS_resetGPU
  (JNIEnv *, jclass);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    calcDataBleachCorrection
 * Signature: ([F[FLgpufitImFCS/GpufitImFCS/ACFParameters;)V
 */
JNIEXPORT void JNICALL Java_gpufitImFCS_GpufitImFCS_calcDataBleachCorrection
  (JNIEnv *, jclass, jfloatArray, jfloatArray, jobject);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    isBinningMemorySufficient
 * Signature: (LgpufitImFCS/GpufitImFCS/ACFParameters;)Z
 */
JNIEXPORT jboolean JNICALL Java_gpufitImFCS_GpufitImFCS_isBinningMemorySufficient
  (JNIEnv *, jclass, jobject);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    calcBinning
 * Signature: ([F[FLgpufitImFCS/GpufitImFCS/ACFParameters;)V
 */
JNIEXPORT void JNICALL Java_gpufitImFCS_GpufitImFCS_calcBinning
  (JNIEnv *, jclass, jfloatArray, jfloatArray, jobject);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    isACFmemorySufficient
 * Signature: (LgpufitImFCS/GpufitImFCS/ACFParameters;)Z
 */
JNIEXPORT jboolean JNICALL Java_gpufitImFCS_GpufitImFCS_isACFmemorySufficient
  (JNIEnv *, jclass, jobject);

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    calcACF
 * Signature: ([F[D[D[D[D[D[D[D[ILgpufitImFCS/GpufitImFCS/ACFParameters;)V
 */
JNIEXPORT void JNICALL Java_gpufitImFCS_GpufitImFCS_calcACF
  (JNIEnv *, jclass, jfloatArray, jdoubleArray, jdoubleArray, jdoubleArray, jdoubleArray, jdoubleArray, jdoubleArray, jdoubleArray, jintArray, jobject);

#ifdef __cplusplus
}
#endif
/* -------------------------------------------------------------------------------------------------------
* from gpufitImFCS_GpufitImFCS.h END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from constants.h START
------------------------------------------------------------------------------------------------------- */
// fitting model ID

enum ModelID {
    GAUSS_2D = 1,
    ACF_1D= 2,
    LINEAR_1D = 3
};

// estimator ID
enum EstimatorID { LSE = 0, MLE = 1 };

// fit state
enum FitState { CONVERGED = 0, MAX_ITERATION = 1, SINGULAR_HESSIAN = 2, NEG_CURVATURE_MLE = 3, GPU_NOT_READY = 4 };

// return state
enum ReturnState { OK = 0, ERROR = -1 };

enum DataLocation { HOST = 0, DEVICE = 1 };
/* -------------------------------------------------------------------------------------------------------
* from constants.h END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from models/gauss_2d.cuh START
------------------------------------------------------------------------------------------------------- */
/* Description of the calculate_gauss2d function
* ==============================================
*
* This function calculates the values of two-dimensional gauss model functions
* and their partial derivatives with respect to the model parameters. 
*
* No independent variables are passed to this model function.  Hence, the 
* (X, Y) coordinate of the first data value is assumed to be (0.0, 0.0).  For
* a fit size of M x N data points, the (X, Y) coordinates of the data are
* simply the corresponding array index values of the data array, starting from
* zero.
*
* Parameters:
*
* parameters: An input vector of model parameters.
*             p[0]: amplitude
*             p[1]: center coordinate x
*             p[2]: center coordinate y
*             p[3]: width (standard deviation; equal width in x and y dimensions)
*             p[4]: offset
*
* n_fits: The number of fits. (not used)
*
* n_points: The number of data points per fit.
*
* value: An output vector of model function values.
*
* derivative: An output vector of model function partial derivatives.
*
* point_index: The data point index.
*
* fit_index: The fit index. (not used)
*
* chunk_index: The chunk index. (not used)
*
* user_info: An input vector containing user information. (not used)
*
* user_info_size: The size of user_info in bytes. (not used)
*
* Calling the calculate_gauss2d function
* ======================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_gauss2d(
    REAL const * parameters,
    int const n_fits,
    int const n_points,
    REAL * value,
    REAL * derivative,
    int const point_index,
    int const fit_index,
    int const chunk_index,
    char * user_info,
    std::size_t const user_info_size,
    int const num_v_coefs) // NEW
{
    // indices

    int const n_points_x = sqrt((REAL)n_points);
    int const point_index_y = point_index / n_points_x;
    int const point_index_x = point_index - point_index_y * n_points_x;

    // parameters

    REAL const * p = parameters;

    // value

    REAL const argx = (point_index_x - p[1]) * (point_index_x - p[1]) / (2 * p[3] * p[3]);
    REAL const argy = (point_index_y - p[2]) * (point_index_y - p[2]) / (2 * p[3] * p[3]);
    REAL const ex = exp(-(argx + argy));
    value[point_index] = p[0] * ex + p[4];

    // derivatives

    REAL * current_derivative = derivative + point_index;

    current_derivative[0 * n_points] = ex;
    current_derivative[1 * n_points] = p[0] * ex * (point_index_x - p[1]) / (p[3] * p[3]);
    current_derivative[2 * n_points] = p[0] * ex * (point_index_y - p[2]) / (p[3] * p[3]);
    current_derivative[3 * n_points] = ex * p[0] * ((point_index_x - p[1]) * (point_index_x - p[1]) + (point_index_y - p[2]) * (point_index_y - p[2])) / (p[3] * p[3] * p[3]);
    current_derivative[4 * n_points] = 1;
}
/* -------------------------------------------------------------------------------------------------------
* from models/gauss_2d.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from models/linear_1d.cuh START
------------------------------------------------------------------------------------------------------- */
/* Description of the calculate_linear1d function
* ===================================================
*
* This function calculates the values of one-dimensional linear model functions
* and their partial derivatives with respect to the model parameters. 
*
* This function makes use of the user information data to pass in the 
* independent variables (X values) corresponding to the data.  The X values
* must be of type REAL.
*
* Note that if no user information is provided, the (X) coordinate of the 
* first data value is assumed to be (0.0).  In this case, for a fit size of 
* M data points, the (X) coordinates of the data are simply the corresponding 
* array index values of the data array, starting from zero.
*
* There are three possibilities regarding the X values:
*
*   No X values provided: 
*
*       If no user information is provided, the (X) coordinate of the 
*       first data value is assumed to be (0.0).  In this case, for a 
*       fit size of M data points, the (X) coordinates of the data are 
*       simply the corresponding array index values of the data array, 
*       starting from zero.
*
*   X values provided for one fit:
*
*       If the user_info array contains the X values for one fit, then 
*       the same X values will be used for all fits.  In this case, the 
*       size of the user_info array (in bytes) must equal 
*       sizeof(REAL) * n_points.
*
*   Unique X values provided for all fits:
*
*       In this case, the user_info array must contain X values for each
*       fit in the dataset.  In this case, the size of the user_info array 
*       (in bytes) must equal sizeof(REAL) * n_points * nfits.
*
* Parameters:
*
* parameters: An input vector of model parameters.
*             p[0]: offset
*             p[1]: slope
*
* n_fits: The number of fits.
*
* n_points: The number of data points per fit.
*
* value: An output vector of model function values.
*
* derivative: An output vector of model function partial derivatives.
*
* point_index: The data point index.
*
* fit_index: The fit index.
*
* chunk_index: The chunk index. Used for indexing of user_info.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the calculate_linear1d function
* =======================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_linear1d(
    REAL const * parameters,
    int const n_fits,
    int const n_points,
    REAL * value,
    REAL * derivative,
    int const point_index,
    int const fit_index,
    int const chunk_index,
    char * user_info,
    std::size_t const user_info_size,
    int const num_v_coefs) // NEW
{
    // indices

    REAL * user_info_float = (REAL*) user_info;
    REAL x = 0;
  //  if (!user_info_float)
  //  {
  //      x = point_index;
  //  }
  //  else if (user_info_size / sizeof(REAL) == n_points)
  //  {
        x = user_info_float[point_index];
   // }
   // else if (user_info_size / sizeof(REAL) > n_points)
  //  {
  //      int const chunk_begin = chunk_index * n_fits * n_points;
  //      int const fit_begin = fit_index * n_points;
  //      x = user_info_float[chunk_begin + fit_begin + point_index];
  //  }

    // value
    //value[point_index] = parameters[0] + parameters[1]*x + parameters[2]*x*x + parameters[3]*x*x*x + parameters[4]*x*x*x*x + parameters[5]*x*x*x*x*x + parameters[6]*x*x*x*x*x*x;
    value[point_index] = parameters[0];    

    // derivatives
    REAL * current_derivatives = derivative + point_index;
    current_derivatives[0 * n_points] = 1;

/*
    REAL tempx = 1.0;
    for (int i = 1; i < num_v_coefs; i++) {
        tempx *= x;
        current_derivatives[i * n_points] = tempx;   
	value[point_index] =value[point_index]+ parameters[i]*tempx;     
    }
*/

    double tempx = 1.0;
    double sumval = (double) parameters[0];
    for (int i = 1; i < num_v_coefs; i++) {
        tempx *= x;
        current_derivatives[i * n_points] = (float) tempx;   
        sumval += (double) parameters[i]*tempx;
    }
    value[point_index] =(float) sumval;     

}
/* -------------------------------------------------------------------------------------------------------
* from models/linear_1d.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from models/acf_1d.cuh START
------------------------------------------------------------------------------------------------------- */
__device__ void calculate_acf1d(
    REAL const * parameters,
    int const n_fits,
    int const n_points,
    REAL * value,
    REAL * derivative,
    int const point_index,
    int const fit_index,
    int const chunk_index,
    char * user_info,
    std::size_t const user_info_size,
    int const num_v_coefs) // NEW
{
    // indices

    REAL * user_info_float = (REAL*) user_info;
    REAL x = 0;
    x = user_info_float[point_index];
    double sqrpi = sqrt((double) 3.14159265359);

    double p0t = sqrt(4 * (double) parameters[1] * x + pow((double) parameters[13], 2.0));
    double p1xt = (double) parameters[11] + (double) parameters[15] - (double) parameters[2] * x;
    double p2xt = (double) parameters[11] - (double) parameters[15] + (double) parameters[2] * x;
    double p3xt = (double) parameters[15] - (double) parameters[2] * x;
    double p4xt = 2 * pow((double) parameters[11], 2.0) + 3 * pow((double) parameters[15], 2.0) - 6 * x * (double) parameters[15] * (double) parameters[2] + 3 * pow(x * (double) parameters[2], 2.0);
    double p5xt = pow(p3xt, 2.0) + pow(p1xt, 2.0);
    double p6xt = pow(p3xt, 2.0) + pow(p2xt, 2.0);
    double p7xt = 2 * (pow((double) parameters[11], 2.0) + pow((double) parameters[15], 2.0) - 2 * x * (double) parameters[15] * (double) parameters[2] + pow(x * (double) parameters[2], 2.0));
    double p1yt = (double) parameters[12] + (double) parameters[16] - (double) parameters[3] * x;
    double p2yt = (double) parameters[12] - (double) parameters[16] + (double) parameters[3] * x;
    double p3yt = (double) parameters[16] - (double) parameters[3] * x;
    double p4yt = 2 * pow((double) parameters[12], 2.0) + 3 * pow((double) parameters[16], 2.0) - 6 * x * (double) parameters[16] * (double) parameters[3] + 3 * pow(x * (double) parameters[3], 2.0);
    double p5yt = pow(p3yt, 2.0) + pow(p1yt, 2.0);
    double p6yt = pow(p3yt, 2.0) + pow(p2yt, 2.0);
    double p7yt = 2 * (pow((double) parameters[12], 2.0) + pow((double) parameters[16], 2.0) - 2 * x * (double) parameters[16] * (double) parameters[3] + pow(x * (double) parameters[3], 2.0));

    double pexpxt = exp(-pow(p1xt / p0t, 2.0)) + exp(-pow(p2xt / p0t, 2.0)) - 2 * exp(-pow(p3xt / p0t, 2.0));
    double perfxt = p1xt * erf(p1xt / p0t) + p2xt * erf(p2xt / p0t) - 2 * p3xt * erf(p3xt / p0t);
    double dDpexpxt = 2 * exp(-p4xt / pow(p0t, 2.0)) * (exp(p5xt / pow(p0t, 2.0)) + exp(p6xt / pow(p0t, 2.0)) - 2 * exp(p7xt / pow(p0t, 2.0)));
    double dvxperfxt = (erf(p2xt / p0t) + 2 * erf(p3xt / p0t) - erf(p1xt / p0t)) * x;
    double pexpyt = exp(-pow(p1yt / p0t, 2.0)) + exp(-pow(p2yt / p0t, 2.0)) - 2 * exp(-pow(p3yt / p0t, 2.0));
    double dDpexpyt = 2 * exp(-p4yt / pow(p0t, 2.0)) * (exp(p5yt / pow(p0t, 2.0)) + exp(p6yt / pow(p0t, 2.0)) - 2 * exp(p7yt / pow(p0t, 2.0)));
    double dvyperfyt = (erf(p2yt / p0t) + 2 * erf(p3yt / p0t) - erf(p1yt / p0t)) * x;
    double perfyt = p1yt * erf(p1yt / p0t) + p2yt * erf(p2yt / p0t) - 2 * p3yt * erf(p3yt / p0t);
    double pplane1 = (p0t / sqrpi * pexpxt + perfxt) * (p0t / sqrpi * pexpyt + perfyt) / (4 * (double) parameters[11]*(double) parameters[12]) * ((double) parameters[17] / ((double) parameters[11]*(double) parameters[12]));
    double pspim1 = 1 / sqrt(1 + (4 * (double) parameters[1] * x) / powf((double) parameters[14], 2));
    double acf1 = pplane1 * pspim1;

    double p0t2 = sqrt(4 * (double) parameters[6] * x + pow((double) parameters[13], 2));
    double p1xt2 = (double) parameters[11] + (double) parameters[15] - (double) parameters[2] * x;
    double p2xt2 = (double) parameters[11] - (double) parameters[15] + (double) parameters[2] * x;
    double p3xt2 = (double) parameters[15] - (double) parameters[2] * x;
    double p4xt2 = 2 * pow((double) parameters[11], 2) + 3 * pow((double) parameters[15], 2) - 6 * x * (double) parameters[15] * (double) parameters[2] + 3 * pow(x * (double) parameters[2], 2);
    double p5xt2 = pow(p3xt2, 2) + pow(p1xt2, 2);
    double p6xt2 = pow(p3xt2, 2) + pow(p2xt2, 2);
    double p7xt2 = 2 * (pow((double) parameters[11], 2) + pow((double) parameters[15], 2) - 2 * x * (double) parameters[15] * (double) parameters[2] + pow(x * (double) parameters[2], 2));
    double p1yt2 = (double) parameters[12] + (double) parameters[16] - (double) parameters[3] * x;
    double p2yt2 = (double) parameters[12] - (double) parameters[16] + (double) parameters[3] * x;
    double p3yt2 = (double) parameters[16] - (double) parameters[3] * x;
    double p4yt2 = 2 * pow((double) parameters[12], 2) + 3 * pow((double) parameters[16], 2) - 6 * x * (double) parameters[16] * (double) parameters[3] + 3 * pow(x * (double) parameters[3], 2);
    double p5yt2 = pow(p3yt2, 2) + pow(p1yt2, 2);
    double p6yt2 = pow(p3yt2, 2) + pow(p2yt2, 2);
    double p7yt2 = 2 * (pow((double) parameters[12], 2) + pow((double) parameters[16], 2) - 2 * x * (double) parameters[16] * (double) parameters[3] + pow(x * (double) parameters[3], 2));
    double pexpxt2 = exp(-pow(p1xt2 / p0t2, 2)) + exp(-pow(p2xt2 / p0t2, 2)) - 2 * exp(-pow(p3xt2 / p0t2, 2));
    double perfxt2 = p1xt2 * erf(p1xt2 / p0t2) + p2xt2 * erf(p2xt2 / p0t2) - 2 * p3xt2 * erf(p3xt2 / p0t2);
    double dDpexpxt2 = 2 * exp(-p4xt2 / pow(p0t2, 2)) * (exp(p5xt2 / pow(p0t2, 2)) + exp(p6xt2 / pow(p0t2, 2)) - 2 * exp(p7xt2 / pow(p0t2, 2)));
    double dvxperfxt2 = (erf(p2xt2 / p0t2) + 2 * erf(p3xt2 / p0t2) - erf(p1xt2 / p0t2)) * x;
    double pexpyt2 = exp(-pow(p1yt2 / p0t2, 2)) + exp(-pow(p2yt2 / p0t2, 2)) - 2 * exp(-pow(p3yt2 / p0t2, 2));
    double dDpexpyt2 = 2 * exp(-p4yt2 / pow(p0t2, 2)) * (exp(p5yt2 / pow(p0t2, 2)) + exp(p6yt2 / powf(p0t2, 2)) - 2 * exp(p7yt2 / powf(p0t2, 2)));
    double dvyperfyt2 = (erf(p2yt2 / p0t2) + 2 * erf(p3yt2 / p0t2) - erf(p1yt2 / p0t2)) * x;
    double perfyt2 = p1yt2 * erf(p1yt2 / p0t2) + p2yt2 * erf(p2yt2 / p0t2) - 2 * p3yt2 * erf(p3yt2 / p0t2);
    double pplane2 = (p0t2 / sqrpi * pexpxt2 + perfxt2) * (p0t2 / sqrpi * pexpyt2 + perfyt2) / (4 * pow((double) parameters[11]*(double) parameters[12], 2) / (double) parameters[17]);
    double pspim2 = 1 / sqrt(1 + (4 * (double) parameters[6] * x) / pow((double) parameters[14], 2));
    double acf2 = pplane2 * pspim2;

    double triplet = 1 + (double) parameters[9] / (1 - (double) parameters[9]) * exp(-x / (double) parameters[10]);

    value[point_index] = (1 / (double) parameters[0]) * ((1 - (double) parameters[5]) * acf1 + powf((double) parameters[18], 2) * (double) parameters[5] * acf2) / pow(1 - (double) parameters[5] + (double) parameters[18] * (double) parameters[5], 2) * triplet + (double) parameters[4];

    double dDplat = (1 / (sqrpi * p0t)) * (dDpexpyt * x * (p0t / sqrpi * pexpxt + perfxt) + dDpexpxt * x * (p0t / sqrpi * pexpyt + perfyt)) / (4 * powf((double) parameters[11]*(double) parameters[12], 2.0) / (double) parameters[17]);
    double dDpspim = -4 * x / (2 * pow((double) parameters[14], 2) * pow(sqrt(1 + (4 * (double) parameters[1] * x) / pow((double) parameters[14], 2)), 3));

    double dDplat2 = (1 / (sqrpi * p0t2)) * (dDpexpyt2 * x * (p0t2 / sqrpi * pexpxt2 + perfxt2) + dDpexpxt2 * x * (p0t2 / sqrpi * pexpyt2 + perfyt2)) / (4 * pow((double) parameters[11]*(double) parameters[12], 2) / (double) parameters[17]);
    double dDpspim2 = -4 * x / (2 * pow((double) parameters[14], 2) * pow(sqrt(1 + (4 * (double) parameters[6] * x) / pow((double) parameters[14], 2)), 3));

    double dtripletFtrip = exp(-x / (double) parameters[10]) * (1 / (1 - (double) parameters[9]) + (double) parameters[9] / pow(1 - (double) parameters[9], 2));
    double dtripletTtrip = exp(-x / (double) parameters[10]) * ((double) parameters[9] * x) / ((1 - (double) parameters[9]) * pow((double) parameters[10], 2));

    double pf1 = (1 - (double) parameters[5]) / (1 - (double) parameters[5] + (double) parameters[18] * (double) parameters[5]);
    double pf2 = (pow((double) parameters[18], 2) * (double) parameters[5]) / (1 - (double) parameters[5] + (double) parameters[18] * (double) parameters[5]);
    double dfnom = pow(1 - (double) parameters[5] + (double) parameters[18] * (double) parameters[5], 3);
    double df21 = 1 - (double) parameters[5] + (double) parameters[18] * (double) parameters[5] - 2 * (double) parameters[18];
    double df22 = pow((double) parameters[18], 2) * (1 + (double) parameters[5] - (double) parameters[18] * (double) parameters[5]);

    double pacf = (1 / (double) parameters[0]) * ((1 - (double) parameters[5]) * acf1 + powf((double) parameters[18], 2) * (double) parameters[5] * acf2) / pow(1 - (double) parameters[5] + (double) parameters[18] * (double) parameters[5], 2) * triplet + (double) parameters[4];

    REAL * current_derivatives = derivative + point_index;

    current_derivatives[0 * n_points] = (float) (-1 / pow((double)parameters[0], 2)) * (pf1 * acf1 + pf2 * acf2) * triplet;
    current_derivatives[1 * n_points] = (1 /  parameters[0]) * (float)(pf1 *  (pplane1 * dDpspim + pspim1 * dDplat));
    current_derivatives[2 * n_points] = (1 /   parameters[0]) * (float)((pf1 * ((p0t / sqrpi * pexpyt + perfyt) * dvxperfxt) * pspim1 / (4 * pow((double) parameters[11]*(double) parameters[12], 2) /  parameters[17]) + pf2 * ((p0t2 / sqrpi * pexpyt2 + perfyt2) * dvxperfxt2) * pspim2 / (4 * pow((double) parameters[11]*(double) parameters[12], 2) /  parameters[17])) * triplet);
    current_derivatives[3 * n_points] = (1 /   parameters[0]) * (float)((pf1 * ((p0t / sqrpi * pexpxt + perfxt) * dvyperfyt) * pspim1 / (4 * pow((double) parameters[11]*(double) parameters[12], 2) /   parameters[17]) + pf2 * ((p0t2 / sqrpi * pexpxt2 + perfxt2) * dvyperfyt2) * pspim2 / (4 * pow((double) parameters[11]*(double) parameters[12], 2) /   parameters[17])) * triplet);
    current_derivatives[4 * n_points] = 1.0;
    current_derivatives[5 * n_points] = (1 /  parameters[0]) *(float)((1 / dfnom) * (df21 * acf1 + df22 * acf2) * triplet);
    current_derivatives[6 * n_points] = (1 /  parameters[0]) * (float)(pf2 * (pplane2 * dDpspim2 + pspim2 * dDplat2) * triplet);
    current_derivatives[9 * n_points] = (float)dtripletFtrip * pacf;
    current_derivatives[10 * n_points] = (float)dtripletTtrip * pacf;
    //current_derivatives[11 * n_points] = 0.0;
    //current_derivatives[12 * n_points] = 0.0;
    //current_derivatives[13 * n_points] = 0.0;
    //current_derivatives[14 * n_points] = 0.0;
    //current_derivatives[15 * n_points] = 0.0;
    //current_derivatives[16 * n_points] = 0.0;
    //current_derivatives[17 * n_points] = 0.0;
    //current_derivatives[18 * n_points] = 0.0;
    //current_derivatives[19 * n_points] = 0.0; 
}
/* -------------------------------------------------------------------------------------------------------
* from models/acf_1d.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from models/models.cuh START
------------------------------------------------------------------------------------------------------- */
__device__ void calculate_model(
    ModelID const model_id,
    REAL const * parameters,
    int const n_fits,
    int const n_points,
    REAL * value,
    REAL * derivative,
    int const point_index,
    int const fit_index,
    int const chunk_index,
    char * user_info,
    int const user_info_size,
    int const num_v_coefs) // NEW
{
    switch (model_id)
    {
    case GAUSS_2D:
        calculate_gauss2d(parameters, n_fits, n_points, value, derivative, point_index, fit_index, chunk_index, user_info, user_info_size, num_v_coefs);
        break;
    case ACF_1D:
	calculate_acf1d(parameters, n_fits, n_points, value, derivative, point_index, fit_index, chunk_index, user_info, user_info_size, num_v_coefs);
	break;
    case LINEAR_1D:
        calculate_linear1d(parameters, n_fits, n_points, value, derivative, point_index, fit_index, chunk_index, user_info, user_info_size, num_v_coefs);
        break;
    default:
        break;
    }
}

void configure_model(ModelID const model_id, int & n_parameters, int & n_dimensions)
{
    switch (model_id)
    {
    case GAUSS_2D:              n_parameters = 5;  n_dimensions = 2;  break;
    case ACF_1D:                n_parameters = 20; n_dimensions = 1;  break;
    case LINEAR_1D:             n_parameters = 11; n_dimensions = 1;  break;
    default:                                                          break;
    }
}
/* -------------------------------------------------------------------------------------------------------
* from models/models.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from estimators/lse.cuh START
------------------------------------------------------------------------------------------------------- */
/* Description of the calculate_chi_square_lse function
* =====================================================
*
* This function calculates the chi-square values for the weighted LSE estimator.
*
* Parameters:
*
* chi_square: An output vector of chi-square values for each data point.
*
* point_index: The data point index.
*
* data: An input vector of data values.
*
* value: An input vector of fitting curve values.
*
* weight: An optional input vector of values for weighting the chi-square values.
*
* state: A pointer to a value which indicates whether the fitting
*        process was carreid out correctly or which problem occurred.
*        In this function it is not used. It can be used in functions calculating
*        other estimators than the LSE, such as MLE. It is passed into this function
*        to provide the same interface for all estimator functions.
*
* user_info: An input vector containing user information. (not used)
*
* user_info_size: The number of elements in user_info. (not used)
*
* Calling the calculate_chi_square_lse function
* =============================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_chi_square_lse(
    volatile REAL * chi_square,
    int const point_index,
    REAL const * data,
    REAL const * value,
    REAL const * weight,
    int * state,
    char * user_info,
    std::size_t const user_info_size)
{
    REAL const deviation = value[point_index] - data[point_index];

    if (weight)
    {
        chi_square[point_index] = deviation * deviation * weight[point_index];
    }
    else
    {
        chi_square[point_index] = deviation * deviation;
    }
}

/* Description of the calculate_hessian_lse function
* ==================================================
*
* This function calculates the hessian matrix values of the weighted LSE estimator.
* The calculation is performed based on previously calculated fitting curve derivative
* values.
*
* Parameters:
*
* hessian: An output vector of values of the hessian matrix for each data point.
*
* point_index: The data point index.
*
* parameter_index_i: Index of the hessian column.
*
* parameter_index_j: Index of the hessian row.
*
* data: An input vector of data values. (not used)
*
* value: An input vector of fitting curve values. (not used)
*
* derivative: An input vector of partial derivative values of the fitting
*             curve with respect to the fitting parameters for each data point.
*
* weight: An optional input vector of values for weighting the hessian matrix values.
*
* user_info: An input vector containing user information. (not used)
*
* user_info_size: The number of elements in user_info. (not used)
*
* Calling the calculate_hessian_lse function
* ==========================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_hessian_lse(
    double * hessian,
    int const point_index,
    int const parameter_index_i,
    int const parameter_index_j,
    REAL const * data,
    REAL const * value,
    REAL const * derivative,
    REAL const * weight,
    char * user_info,
    std::size_t const user_info_size)
{
    if (weight)
    {
        *hessian
            += derivative[parameter_index_i] * derivative[parameter_index_j]
            * weight[point_index];
    }
    else
    {
        *hessian
            += derivative[parameter_index_i] * derivative[parameter_index_j];
    }
}

/* Description of the calculate_gradient_lse function
* ===================================================
*
* This function calculates the gradient values of the weighted LSE estimator
* based on previously calculated fitting curve derivative values.
*
* Parameters:
*
* gradient: An output vector of values of the gradient vector for each data point.
*
* point_index: The data point index.
*
* parameter_index: The parameter index.
*
* n_parameters: The number of fitting curve parameters.
*
* data: An input vector of data values.
*
* value: An input vector of fitting curve values.
*
* derivative: An input vector of partial derivative values of the fitting
*             curve with respect to the fitting parameters for each data point.
*
* weight: An optional input vector of values for weighting gradient values.
*
* user_info: An input vector containing user information. (not used)
*
* user_info_size: The number of elements in user_info. (not used)
*
* Calling the calculate_gradient_lse function
* ===========================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_gradient_lse(
    volatile REAL * gradient,
    int const point_index,
    int const parameter_index,
    REAL const * data,
    REAL const * value,
    REAL const * derivative,
    REAL const * weight,
    char * user_info,
    std::size_t const user_info_size)
{
    REAL const deviation = data[point_index] - value[point_index];

    if (weight)
    {
        gradient[point_index]
            = derivative[parameter_index] * deviation * weight[point_index];
    }
    else
    {
        gradient[point_index]
            = derivative[parameter_index] * deviation;
    }
}
/* -------------------------------------------------------------------------------------------------------
* from estimators/lse.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from estimators/mle.cuh START
------------------------------------------------------------------------------------------------------- */
/* Description of the calculate_chi_square_mle function
* =====================================================
*
* This function calculates the chi-square values for the MLE estimator.
*
* Parameters:
*
* chi_square: An output vector of chi-square values for each data point.
*
* point_index: The data point index.
*
* data: An input vector of data.
*
* value: An input vector of fitting curve values.
*
* weight: An input vector of values for weighting chi-square values. It is not used
*         in this function. It can be used in functions calculating other estimators
*         than the MLE, such as LSE.
*
* state: A pointer to a value which indicates whether the fitting process was carreid
*        out correctly or which problem occurred. It is set to 3 if a fitting curve
*        value is negative.
*
* user_info: An input vector containing user information. (not used)
*
* user_info_size: The number of elements in user_info. (not used)
*
* Calling the calculate_chi_square_mle function
* =============================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_chi_square_mle(
    volatile REAL * chi_square,
    int const point_index,
    REAL const * data,
    REAL const * value,
    REAL const * weight,
    int * state,
    char * user_info,
    std::size_t const user_info_size)
{
    if (value[point_index] < 0)
    {
        *state = 3;
    }

    REAL const deviation = value[point_index] - data[point_index];

    if (data[point_index] != 0)
    {
        chi_square[point_index]
            = 2 * (deviation - data[point_index] * std::log(value[point_index] / data[point_index]));
    }
    else
    {
        chi_square[point_index] = 2 * deviation;
    }
}

/* Description of the calculate_hessian_mle function
* ==================================================
*
* This function calculates the hessian matrix values of the MLE estimator. The
* calculation is performed based on previously calculated derivative values.
* 
* Parameters:
*
* hessian: An output vector of values of the hessian matrix for each data point.
*
* point_index: The data point index.
*
* parameter_index_i: Index of the hessian column.
*
* parameter_index_j: Index of the hessian row.
*
* data: An input vector of data values.
*
* value: An input vector of fitting curve values.
*
* derivative: An input vector of partial derivative values of the fitting
*             curve with respect to the fitting parameters for each data point.
*
* weight: An input vector of values for weighting hessian matrix values. It is not
*         used in this function. It can be used in functions calculating other estimators
*         than the MLE, such as LSE.
*
* user_info: An input vector containing user information. (not used)
*
* user_info_size: The number of elements in user_info. (not used)
*
* Calling the calculate_hessian_mle function
* ==========================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_hessian_mle(
    double * hessian,
    int const point_index,
    int const parameter_index_i,
    int const parameter_index_j,
    REAL const * data,
    REAL const * value,
    REAL const * derivative,
    REAL const * weight,
    char * user_info,
    std::size_t const user_info_size)
{
    *hessian
        += data[point_index]
        / (value[point_index] * value[point_index])
        * derivative[parameter_index_i] * derivative[parameter_index_j];
}

/* Description of the calculate_gradient_mle function
* ===================================================
*
* This function calculates the gradient values of the MLE estimator based
* on previously calculated derivative values.
*
* Parameters:
*
* gradient: An output vector of values of the gradient vector for each data point.
*
* point_index: The data point index.
*
* parameter_index: The parameter index.
*
* data: An input vector of data values.
*
* value: An input vector of fitting curve values.
*
* derivative: An input vector of partial derivative values of the fitting
*             curve with respect to the fitting parameters for each data point.
*
* weight: An input vector of values for weighting gradient vector values. It is not
*         used in this function. It can be used in functions calculating other estimators
*         than the MLE, such as LSE.
*
* user_info: An input vector containing user information. (not used)
*
* user_info_size: The number of elements in user_info. (not used)
*
* Calling the calculate_gradient_mle function
* ===========================================
*
* This __device__ function can be only called from a __global__ function or an other
* __device__ function.
*
*/

__device__ void calculate_gradient_mle(
    volatile REAL * gradient,
    int const point_index,
    int const parameter_index,
    REAL const * data,
    REAL const * value,
    REAL const * derivative,
    REAL const * weight,
    char * user_info,
    std::size_t const user_info_size)
{
    gradient[point_index]
        = -derivative[parameter_index]
        * (1 - data[point_index] / value[point_index]);
}
/* -------------------------------------------------------------------------------------------------------
* from estimators/mle.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from estimators/estimators.cuh START
------------------------------------------------------------------------------------------------------- */
__device__ void calculate_chi_square(
    int const estimator_id,
    volatile REAL * chi_square,
    int const point_index,
    REAL const * data,
    REAL const * value,
    REAL const * weight,
    int * state,
    char * user_info,
    std::size_t const user_info_size)
{
    switch (estimator_id)
    {
    case LSE:
        calculate_chi_square_lse(chi_square, point_index, data, value, weight, state, user_info, user_info_size);
        break;
    case MLE:
        calculate_chi_square_mle(chi_square, point_index, data, value, weight, state, user_info, user_info_size);
        break;
    default:
        break;
    }
}

__device__ void calculate_gradient(
    int const estimator_id,
    volatile REAL * gradient,
    int const point_index,
    int const parameter_index,
    REAL const * data,
    REAL const * value,
    REAL const * derivative,
    REAL const * weight,
    char * user_info,
    std::size_t const user_info_size)
{
    switch (estimator_id)
    {
    case LSE:
        calculate_gradient_lse(gradient, point_index, parameter_index, data, value, derivative, weight, user_info, user_info_size);
        break;
    case MLE:
        calculate_gradient_mle(gradient, point_index, parameter_index, data, value, derivative, weight, user_info, user_info_size);
        break;
    default:
        break;
    }
}

__device__ void calculate_hessian(
    int const estimator_id,
    double * hessian,
    int const point_index,
    int const parameter_index_i,
    int const parameter_index_j,
    REAL const * data,
    REAL const * value,
    REAL const * derivative,
    REAL const * weight,
    char * user_info,
    std::size_t const user_info_size)
{
    switch (estimator_id)
    {
    case LSE:
        calculate_hessian_lse
        (hessian, point_index, parameter_index_i, parameter_index_j, data, value, derivative, weight, user_info,user_info_size);
        break;
    case MLE:
        calculate_hessian_mle
        (hessian, point_index, parameter_index_i, parameter_index_j, data, value, derivative, weight, user_info, user_info_size);
        break;
    default:
        break;
    }
}
/* -------------------------------------------------------------------------------------------------------
* from estimators/estimators.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from cuda_kernel.cuh START
------------------------------------------------------------------------------------------------------- */
/*
void configure_model(ModelID const model_id, int & n_parameters, int & n_dimensions);

extern __global__ void cuda_sum_chi_square_subtotals(
    REAL * chi_squares,
    REAL const * subtotals,
    int const n_blocks_per_fit,
    int const n_fits,
    int const * finished);

extern __global__ void cuda_check_fit_improvement(
    int * iteration_failed,
    REAL const * chi_squares,
    REAL const * prev_chi_squares,
    int const n_fits,
    int const * finished);

extern __global__ void cuda_calculate_chi_squares(
    REAL * chi_squares,
    int * states,
    REAL const * data,
    REAL const * values,
    REAL const * weights,
    int const n_points,
    int const n_fits,
    int const estimator_id,
    int const * finished,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size);

extern __global__ void cuda_sum_gradient_subtotals(
    REAL * gradients,
    REAL const * subtotals,
    int const n_blocks_per_fit,
    int const n_fits,
    int const n_parameters,
    int const * skip,
    int const * finished);

extern __global__ void cuda_calculate_gradients(
    REAL * gradients,
    REAL const * data,
    REAL const * values,
    REAL const * derivatives,
    REAL const * weights,
    int const n_points,
    int const n_fits,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
	int const estimator_id,
    int const * finished,
    int const * skip,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size);

extern __global__ void cuda_calculate_hessians(
    REAL * hessians,
    REAL const * data,
    REAL const * values,
    REAL const * derivatives,
    REAL const * weights,
    int const n_fits,
    int const n_points,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const estimator_id,
    int const * skip,
    int const * finished,
    char * user_info,
    std::size_t const user_info_size);

extern __global__ void cuda_modify_step_widths(
    REAL * hessians,
    REAL const * lambdas,
    REAL * scaling_vectors,
    unsigned int const n_parameters,
    int const * iteration_failed,
    int const * finished,
    int const n_fits_per_block);

extern __global__ void cuda_calc_curve_values(
    REAL const * parameters,
    int const n_fits,
    int const n_points,
    int const n_parameters,
    int const * finished,
    REAL * values,
    REAL * derivatives,
    int const n_fits_per_block,
    int const n_blocks_per_fit,
    ModelID const model_id,
    int const chunk_index,
    char * user_info,
    std::size_t const user_info_size,
    int const num_v_coefs); // NEW

extern __global__ void cuda_update_parameters(
    REAL * parameters,
    REAL * prev_parameters,
    REAL const * deltas,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const * finished,
    int const n_fits_per_block);

extern __global__ void cuda_check_for_convergence(
    int * finished,
    REAL const tolerance,
    int * states,
    REAL const * chi_squares,
    REAL const * prev_chi_squares,
    int const iteration,
    int const max_n_iterations,
    int const n_fits);

extern __global__ void cuda_evaluate_iteration(
    int * all_finished,
    int * n_iterations,
    int * finished,
    int const iteration,
    int const * states,
    int const n_fits);

extern __global__ void cuda_prepare_next_iteration(
    REAL * lambdas,
    REAL * chi_squares,
    REAL * prev_chi_squares,
    REAL * function_parameters,
    REAL const * prev_parameters,
    int const n_fits,
    int const n_parameters);

extern __global__ void cuda_update_state_after_solving(
    int const n_fits,
    int const * singular_checks,
    int const * finished,
    int * states);
*/
/* -------------------------------------------------------------------------------------------------------
* from cuda_kernel.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from cuda_kernel.cu START
------------------------------------------------------------------------------------------------------- */
/* Description of the cuda_calc_curve_values function
* ===================================================
*
* This function calls one of the fitting curve functions depending on the input
* parameter model_id. The fitting curve function calculates the values of
* the fitting curves and its partial derivatives with respect to the fitting
* curve parameters. Multiple fits are calculated in parallel.
*
* Parameters:
*
* parameters: An input vector of concatenated sets of model parameters.
*
* n_fits: The number of fits.
*
* n_points: The number of data points per fit.
*
* n_parameters: The number of curve parameters.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* values: An output vector of concatenated sets of model function values.
*
* derivatives: An output vector of concatenated sets of model function partial
*              derivatives.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* n_blocks_per_fit: The number of thread blocks used to calculate one fit.
*
* model_id: The fitting model ID.
*
* chunk_index: The data chunk index.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calc_curve_values function
* ===========================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   cuda_calc_curve_values<<< blocks, threads >>>(
*       parameters,
*       n_fits,
*       n_points,
*       n_parameters,
*       finished,
*       values,
*       derivatives,
*       n_fits_per_block,
*       n_blocks_per_fit,
*       model_id,
*       chunk_index,
*       user_info,
*       user_info_size,
        num_v_coefs);
*
*/

__global__ void cuda_calc_curve_values(
    REAL const * parameters,
    int const n_fits,
    int const n_points,
    int const n_parameters,
    int const * finished,
    REAL * values,
    REAL * derivatives,
    int const n_fits_per_block,
    int const n_blocks_per_fit,
    ModelID const model_id,
    int const chunk_index,
    char * user_info,
    std::size_t const user_info_size,
    int const num_v_coefs) // NEW
{
    int const fit_in_block = threadIdx.x / n_points;
    int const fit_index = blockIdx.x * n_fits_per_block / n_blocks_per_fit + fit_in_block;
    int const fit_piece = blockIdx.x % n_blocks_per_fit;
    int const point_index = threadIdx.x - fit_in_block * n_points + fit_piece * blockDim.x;
    int const first_point = fit_index * n_points;

    REAL * current_values = values + first_point;
    REAL * current_derivatives = derivatives + first_point * n_parameters;
    REAL const * current_parameters = parameters + fit_index * n_parameters;

    if (finished[fit_index])
        return;
    if (point_index >= n_points)
        return;

    calculate_model(model_id, current_parameters, n_fits, n_points, current_values, current_derivatives, point_index, fit_index, chunk_index, user_info, user_info_size, num_v_coefs);
}

/* Description of the sum_up_floats function
* ==========================================
*
* This function sums up a vector of REAL values and stores the result at the
* first place of the vector.
*
* Parameters:
*
* shared_array: An input vector of REAL values. The vector must be stored
*               on the shared memory of the GPU. The size of this vector must be a
*               power of two. Use zero padding to extend it to the next highest
*               power of 2 greater than the number of elements.
*
* size: The number of elements in the input vector considering zero padding.
*
* Calling the sum_up_floats function
* ==================================
*
* This __device__ function can be only called from a __global__ function or
* an other __device__ function. When calling the function, the blocks and threads
* of the __global__ function must be set up correctly, as shown in the following
* example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = size * vectors_per_block;
*   blocks.x = n_vectors / vectors_per_block;
*
*   global_function<<< blocks, threads >>>(parameter1, ...);
*
*/

__device__ void sum_up_floats(volatile REAL* shared_array, int const size)
{
    int const fit_in_block = threadIdx.x / size;
    int const point_index = threadIdx.x - (fit_in_block*size);

    int current_n_points = size >> 1;
    __syncthreads();
    while (current_n_points)
    {
        if (point_index < current_n_points)
        {
            shared_array[point_index] += shared_array[point_index + current_n_points];
        }
        current_n_points >>= 1;
        __syncthreads();
    }
}

/* Description of the cuda_sum_chi_square_subtotals function
* ==========================================================
*
* This function sums up chi_square subtotals in place.
*
* Parameters:
*
* chi_squares: A vector of chi-square values for multiple fits.
*              in: subtotals
*              out: totals
*
* n_blocks_per_fit: The number of blocks used to calculate one fit. It is
*                   equivalent to the number of subtotals per fit.
*
* n_fits: The number of fits.
*
* finished: An input vector which allows the calculation to be skipped
*           for single fits.
*
* Calling the cuda_sum_chi_square_subtotals function
* ==================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_sum_chi_square_subtotals<<< blocks, threads >>>(
*       chi_squares,
*       n_blocks_per_fit,
*       n_fits,
*       finished);
*
*/

__global__ void cuda_sum_chi_square_subtotals(
    REAL * chi_squares,
    REAL const * subtotals,
    int const n_blocks_per_fit,
    int const n_fits,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n_fits || finished[index])
        return;

    REAL * chi_square = chi_squares + index;
    REAL const * subtotal = subtotals + index;

    double sum = 0.0;
    for (int i = 0; i < n_blocks_per_fit; i++)
        sum += subtotal[i * n_fits];

    chi_square[0] = sum;
}

/* Description of the cuda_check_fit_improvement function
* =======================================================
*
* This function checks after each calculation of chi-square values whether the
* currently calculated chi-square values are lower than chi-square values calculated
* in the previous iteration and sets the iteration_failed flags.
*
* Parameters:
*
* iteration_failed: An output vector of flags which indicate whether the fitting
*                   process improved the fit in the last iteration. If yes it is set
*                   to 0 otherwise to 1.
*
* chi_squares: An input vector of chi-square values for multiple fits.
*
* prev_chi_squares: An input vector of chi-square values for multiple fits calculated
*                   in the previous iteration.
*
* n_fits: The number of fits.
*
* finished: An input vector which allows the calculation to be skipped
*           for single fits.
*
* Calling the cuda_check_fit_improvement function
* ===============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_check_fit_improvement <<< blocks, threads >>>(
*       iteration_failed,
*       chi_squares,
*       prev_chi_squares,
*       n_fits,
*       finished);
*
*/

__global__ void cuda_check_fit_improvement(
    int * iteration_failed,
    REAL const * chi_squares,
    REAL const * prev_chi_squares,
    int const n_fits,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n_fits || finished[index])
        return;

    bool const prev_chi_squares_initialized = prev_chi_squares[index] != 0.;
    // chi_squares[index] can be NaN which compares to false with any other number
    bool const chi_square_decreased = (chi_squares[index] < prev_chi_squares[index]);
    if (prev_chi_squares_initialized && !chi_square_decreased)
    {
        iteration_failed[index] = 1;
    }
    else
    {
        iteration_failed[index] = 0;
    }
}

/* Description of the cuda_calculate_chi_squares function
* ========================================================
*
* This function calls one of the estimator funktions depending on the input
* parameter estimator_id. The estimator function calculates the chi-square values.
* The calcluation is performed for multiple fits in parallel.
*
* Parameters:
*
* chi_squares: An output vector of concatenated chi-square values.
*
* states: An output vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred. In this function
*         it is only used for MLE. It is set to 3 if a fitting curve value is
*         negative. This vector includes the states for multiple fits.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_points: The number of data points per fit.
*
* n_fits: The number of fits.
*
* estimator_id: The estimator ID.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calculate_chi_squares function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = power_of_two_n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   int const shared_size = sizeof(REAL) * threads.x;
*
*   cuda_calculate_chi_squares<<< blocks, threads, shared_size >>>(
*       chi_squares,
*       states,
*       data,
*       values,
*       weights,
*       n_points,
*       n_fits,
*       estimator_id,
*       finished,
*       n_fits_per_block,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_chi_squares(
    REAL * chi_squares,
    int * states,
    REAL const * data,
    REAL const * values,
    REAL const * weights,
    int const n_points,
    int const n_fits,
    int const estimator_id,
    int const * finished,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const fit_piece = blockIdx.x / n_fits;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block - fit_piece * n_fits;
    int const point_index = threadIdx.x - fit_in_block * shared_size + fit_piece * shared_size;
    int const first_point = fit_index * n_points;

    if (finished[fit_index])
    {
        return;
    }

    REAL const * current_data = &data[first_point];
    REAL const * current_weight = weights ? &weights[first_point] : NULL;
    REAL const * current_value = &values[first_point];
    int * current_state = &states[fit_index];

    extern __shared__ REAL extern_array[];

    volatile REAL * shared_chi_square
        = extern_array + (fit_in_block - fit_piece) * shared_size;

    if (point_index >= n_points)
    {
        shared_chi_square[point_index] = 0.;
    }

    if (point_index < n_points)
    {
        calculate_chi_square(
            estimator_id,
            shared_chi_square,
            point_index,
            current_data,
            current_value,
            current_weight,
            current_state,
            user_info,
            user_info_size);
    }
    shared_chi_square += fit_piece * shared_size;
    sum_up_floats(shared_chi_square, shared_size);
    chi_squares[fit_index + fit_piece * n_fits] = shared_chi_square[0];
}

/* Description of the cuda_sum_gradient_subtotals function
* ========================================================
*
* This function sums up the chi-square gradient subtotals in place.
*
* Parameters:
*
* gradients: A vector of gradient values for multiple fits.
*            in: subtotals
*            out: totals
*
* n_blocks_per_fit: The number of blocks used to calculate one fit
*
* n_fits: The number of fits.
*
* n_parameters_to_fit: The number of model parameters, that are not held fixed.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* Calling the cuda_sum_gradient_subtotals function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_sum_gradient_subtotals<<< blocks,threads >>>(
*       gradients,
*       n_blocks_per_fit,
*       n_fits,
*       n_parameters_to_fit,
*       skip,
*       finished);
*
*/

__global__ void cuda_sum_gradient_subtotals(
    REAL * gradients,
    REAL const * subtotals,
    int const n_blocks_per_fit,
    int const n_fits,
    int const n_parameters,
    int const * skip,
    int const * finished)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;
    int const fit_index = index / n_parameters;

    if (fit_index >= n_fits || finished[fit_index] || skip[fit_index])
        return;

    REAL * gradient = gradients + index;
    REAL const * subtotal = subtotals + index;

    double sum = 0.0;
    for (int i = 0; i < n_blocks_per_fit; i++)
        sum += subtotal[i * n_fits * n_parameters];

    gradient[0] = sum;
}

/* Description of the cuda_calculate_gradients function
* =====================================================
*
* This function calls one of the gradient functions depending on the input
* parameter estimator_id. The gradient function calculates the gradient values
* of the chi-square function calling a __device__ function. The calcluation is
* performed for multiple fits in parallel.
*
* Parameters:
*
* gradients: An output vector of concatenated sets of gradient vector values.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* derivatives: An input vector of concatenated sets of model function partial
*              derivatives.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_points: The number of data points per fit.
*
* n_fits: The number of fits.
*
* n_parameters: The number of fitting curve parameters.
*
* n_parameters_to_fit: The number of fitting curve parameters, that are not held
*                      fixed.
*
* parameters_to_fit_indices: An input vector of indices of fitting curve parameters,
*                            that are not held fixed.
*
* estimator_id: The estimator ID.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* user_info: An input vector containing user information.
*
* user_info_size: The number of elements in user_info.
*
* Calling the cuda_calculate_gradients function
* =============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = power_of_two_n_points * n_fits_per_block / n_blocks_per_fit;
*   blocks.x = n_fits / n_fits_per_block * n_blocks_per_fit;
*
*   int const shared_size = sizeof(REAL) * threads.x;
*
*   cuda_calculate_gradients<<< blocks, threads, shared_size >>>(
*       gradients,
*       data,
*       values,
*       derivatives,
*       weight,
*       n_points,
*       n_fits,
*       n_parameters,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       estimator_id,
*       finished,
*       skip,
*       n_fits_per_block,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_gradients(
    REAL * gradients,
    REAL const * data,
    REAL const * values,
    REAL const * derivatives,
    REAL const * weights,
    int const n_points,
    int const n_fits,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const estimator_id,
    int const * finished,
    int const * skip,
    int const n_fits_per_block,
    char * user_info,
    std::size_t const user_info_size)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const fit_piece = blockIdx.x / n_fits;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block - fit_piece * n_fits;
    int const point_index = threadIdx.x - fit_in_block * shared_size + fit_piece * shared_size;
    int const first_point = fit_index * n_points;

    if (finished[fit_index] || skip[fit_index])
    {
        return;
    }

    REAL const * current_data = &data[first_point];
    REAL const * current_weight = weights ? &weights[first_point] : NULL;
    REAL const * current_derivative = &derivatives[first_point * n_parameters];
    REAL const * current_value = &values[first_point];

    extern __shared__ REAL extern_array[];

    volatile REAL * shared_gradient = extern_array + (fit_in_block - fit_piece) * shared_size;

    if (point_index >= n_points)
    {
        shared_gradient[point_index] = 0.;
    }

    for (int parameter_index = 0; parameter_index < n_parameters_to_fit; parameter_index++)
    {
        if (point_index < n_points)
        {
            int const derivative_index = parameters_to_fit_indices[parameter_index] * n_points + point_index;

            calculate_gradient(
                estimator_id,
                shared_gradient,
                point_index,
                derivative_index,
                current_data,
                current_value,
                current_derivative,
                current_weight,
                user_info,
                user_info_size);
        }
        sum_up_floats(shared_gradient + fit_piece * shared_size, shared_size);
        gradients[(fit_index * n_parameters_to_fit + parameter_index) + fit_piece * n_fits * n_parameters_to_fit]
            = shared_gradient[fit_piece * shared_size];
    }
}

/* Description of the cuda_calculate_hessians function
* ====================================================
*
* This function calls one of the hessian function depending on the input
* parameter estimator_id. The hessian funcion calculates the hessian matrix
* values of the chi-square function calling a __device__ functions. The
* calcluation is performed for multiple fits in parallel.
*
* Parameters:
*
* hessians: An output vector of concatenated sets of hessian matrix values.
*
* data: An input vector of data for multiple fits
*
* values: An input vector of concatenated sets of model function values.
*
* derivatives: An input vector of concatenated sets of model function partial
*              derivatives.
*
* weights: An input vector of values for weighting chi-square, gradient and hessian,
*          while using LSE
*
* n_fits: The number of fits.
*
* n_points: The number of data points per fit.
*
* n_parameters: The number of fitting curve parameters.
*
* n_parameters_to_fit: The number of fitting curve parameters, that are not held
*                      fixed.
*
* parameters_to_fit_indices: An input vector of indices of fitting curve parameters,
*                            that are not held fixed.
*
* estimator_id: The estimator ID.
*
* skip: An input vector which allows the calculation to be skipped for single fits.
*
* finished: An input vector which allows the calculation to be skipped for single
*           fits.
*
* user_info: An input vector containing user information.
*
* user_info_size: The size of user_info in bytes.
*
* Calling the cuda_calculate_hessians function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int n_unique_values = n_parameters_to_fit * (n_parameters_to_fit + 1) / 2;
*
*   threads.x
*       = min(n_unique_values * n_fits_per_block, max_threads_per_block);
*
*   blocks.y
*       = threads.x / max_threads_per_block
*       + int((threads.x % max_threads_per_block) > 0);
*
*   blocks.x
*       = n_fits / n_fits_per_block
*       + int((n_fits % n_fits_per_block) > 0);
*
*   cuda_calculate_hessians<<< blocks, threads >>>(
*       hessians,
*       data,
*       values,
*       derivatives,
*       weight,
*       n_fits,
*       n_points,
*       n_parameters,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       estimator_id,
*       skip,
*       finished,
*       user_info,
*       user_info_size);
*
*/

__global__ void cuda_calculate_hessians(
    REAL * hessians,
    REAL const * data,
    REAL const * values,
    REAL const * derivatives,
    REAL const * weights,
    int const n_fits,
    int const n_points,
    int const n_parameters,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const estimator_id,
    int const * skip,
    int const * finished,
    char * user_info,
    std::size_t const user_info_size)
{
    int const n_unique_values = n_parameters_to_fit * (n_parameters_to_fit + 1) / 2;
    int const n_fits_per_block = blockDim.x * gridDim.y / n_unique_values;
    
    int const fit_in_block
        = (gridDim.y == 1)
        ? (blockIdx.y * blockDim.x + threadIdx.x) / n_unique_values
        : 0;

    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    if (fit_index >= n_fits || finished[fit_index] || skip[fit_index])
    {
        return;
    }

    int const first_point = fit_index * n_points;
    int const parameter_index = (blockIdx.y * blockDim.x + threadIdx.x) - fit_in_block * n_unique_values;

    if (parameter_index >= n_unique_values)
    {
        return;
    }

    int const parameter_index_i
        = n_parameters_to_fit
        - 1.
        - std::floor(
            .5*(
                std::sqrt(
                    - 8. * (parameter_index - n_parameters_to_fit)
                    + 4. * n_parameters_to_fit * (n_parameters_to_fit - 1.)
                    - 7.
                ) - 1.
            )
        );

    int const parameter_index_j
        = parameter_index
        + parameter_index_i
        - parameter_index_i*(n_parameters_to_fit - (parameter_index_i - 1) / 2.);

    REAL * current_hessian = &hessians[fit_index * n_parameters_to_fit * n_parameters_to_fit];
    REAL const * current_data = &data[first_point];
    REAL const * current_weight = weights ? &weights[first_point] : NULL;
    REAL const * current_derivative = &derivatives[first_point*n_parameters];
    REAL const * current_value = &values[first_point];

    int const hessian_index_ij = parameter_index_i * n_parameters_to_fit + parameter_index_j;
    int const hessian_index_ji = parameter_index_j * n_parameters_to_fit + parameter_index_i;
    int const derivative_index_i = parameters_to_fit_indices[parameter_index_i] * n_points;
    int const derivative_index_j = parameters_to_fit_indices[parameter_index_j] * n_points;

    double sum = 0.0;
    for (int point_index = 0; point_index < n_points; point_index++)
    {
        calculate_hessian(
            estimator_id,
            &sum,
            point_index,
            derivative_index_i + point_index,
            derivative_index_j + point_index,
            current_data,
            current_value,
            current_derivative,
            current_weight,
            user_info,
            user_info_size);
    }
    current_hessian[hessian_index_ij] = sum;
    current_hessian[hessian_index_ji] = sum;
}

/* Description of the cuda_modify_step_widths function
* ====================================================
*
* This function midifies the diagonal elements of the hessian matrices by multiplying
* them by the factor (1+ lambda). This operation controls the step widths of the
* iteration. If the last iteration failed, befor modifying the hessian, the diagonal
* elements of the hessian are calculated back to represent unmodified values.
*
* hessians: An input and output vector of hessian matrices, which are modified by
*           the lambda values.
*
* lambdas: An input vector of values for modifying the hessians.
*
* n_parameters: The number of fitting curve parameters.
*
* iteration_failed: An input vector which indicates whether the previous iteration
*                   failed.
*
* finished: An input vector which allows the calculation to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each thread block.
*
* Calling the cuda_modify_step_widths function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_parameters_to_fit * n_fits_per_block;
*   blocks.x = n_fits / n_fits_per_block;
*
*   cuda_modify_step_width<<< blocks, threads >>>(
*       hessians,
*       lambdas,
*       n_parameters,
*       iteration_failed,
*       finished,
*       n_fits_per_block);
*
*/

__global__ void cuda_modify_step_widths(
    REAL * hessians,
    REAL const * lambdas,
    REAL * scaling_vectors,
    unsigned int const n_parameters,
    int const * iteration_failed,
    int const * finished,
    int const n_fits_per_block)
{
    int const shared_size = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / shared_size;
    int const parameter_index = threadIdx.x - fit_in_block * shared_size;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    if (finished[fit_index])
    {
        return;
    }

    REAL * hessian = &hessians[fit_index * n_parameters * n_parameters];
    REAL * scaling_vector = &scaling_vectors[fit_index * n_parameters];
    REAL const & lambda = lambdas[fit_index];

    int const diagonal_index = parameter_index * n_parameters + parameter_index;

    if (iteration_failed[fit_index])
    {
        hessian[diagonal_index] -= scaling_vector[parameter_index] * lambda / 10.;
    }

    // adaptive scaling
    scaling_vector[parameter_index]
        = max(scaling_vector[parameter_index], hessian[diagonal_index]);

    // continuous scaling
    //scaling_vector[parameter_index] = hessian[diagonal_index];
    
    // initial scaling
    //if (scaling_vector[parameter_index] == 0.)
    //    scaling_vector[parameter_index] = hessian[diagonal_index];

    hessian[diagonal_index] += scaling_vector[parameter_index] * lambda;
}

/* Description of the cuda_update_parameters function
* ===================================================
*
* This function stores the fitting curve parameter values in prev_parameters and
* updates them after each iteration.
*
* Parameters:
*
* parameters: An input and output vector of concatenated sets of model
*             parameters.
*
* prev_parameters: An input and output vector of concatenated sets of model
*                  parameters calculated by the previous iteration.
*
* deltas: An input vector of concatenated delta values, which are added to the
*         model parameters.
*
* n_parameters_to_fit: The number of fitted curve parameters.
*
* parameters_to_fit_indices: The indices of fitted curve parameters.
*
* finished: An input vector which allows the parameter update to be skipped for single fits.
*
* n_fits_per_block: The number of fits calculated by each threadblock.
*
* Calling the cuda_update_parameters function
* ===========================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_parameters * n_fits_per_block;
*   blocks.x = n_fits / n_fits_per_block;
*
*   cuda_update_parameters<<< blocks, threads >>>(
*       parameters,
*       prev_parameters,
*       deltas,
*       n_parameters_to_fit,
*       parameters_to_fit_indices,
*       finished,
*       n_fits_per_block);
*
*/

__global__ void cuda_update_parameters(
    REAL * parameters,
    REAL * prev_parameters,
    REAL const * deltas,
    int const n_parameters_to_fit,
    int const * parameters_to_fit_indices,
    int const * finished,
    int const n_fits_per_block)
{
    int const n_parameters = blockDim.x / n_fits_per_block;
    int const fit_in_block = threadIdx.x / n_parameters;
    int const parameter_index = threadIdx.x - fit_in_block * n_parameters;
    int const fit_index = blockIdx.x * n_fits_per_block + fit_in_block;

    REAL * current_parameters = &parameters[fit_index * n_parameters];
    REAL * current_prev_parameters = &prev_parameters[fit_index * n_parameters];

    current_prev_parameters[parameter_index] = current_parameters[parameter_index];

    if (finished[fit_index])
    {
        return;
    }

    if (parameter_index >= n_parameters_to_fit)
    {
        return;
    }

    REAL const * current_deltas = &deltas[fit_index * n_parameters_to_fit];

    current_parameters[parameters_to_fit_indices[parameter_index]] += current_deltas[parameter_index];
}

/* Description of the cuda_update_state_after_solving function
 * ===========================================================
 *
 * This function interprets the singular flag vector of the equation system
 * solving function according to this LM implementation.
 *
 * Parameters:
 *
 * n_fits: The number of fits.
 *
 * solution_info: An input vector used to report whether a fit is singular.
 *
 * finished: An input vector which allows the calculation to by skipped for
 *           single fits.
 *
 * gpufit_states: An output vector of values which indicate whether the fitting
 *                process was carreid out correctly or which problem occurred.
 *                If a hessian matrix of a fit is singular, it is set to 2.
 *
 * Calling the cuda_update_state_after_solving function
 * ====================================================
 *
 * When calling the function, the blocks and threads must be set up correctly,
 * as shown in the following example code.
 *
 *   dim3  threads(1, 1, 1);
 *   dim3  blocks(1, 1, 1);
 *
 *   int const example_value = 256;
 *
 *   threads.x = min(n_fits, example_value);
 *   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
 *
 *   cuda_update_state_after_solving<<< blocks, threads >>>(
 *       n_fits,
 *       solution_info,
 *       finished,
 *       gpufit_states);
 *
 */
    
__global__ void cuda_update_state_after_solving(
    int const n_fits,
    int const * cublas_info,
    int const * finished,
    int * states)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
        return;

    if (finished[fit_index])
        return;

    if (cublas_info[fit_index] != 0)
        states[fit_index] = SINGULAR_HESSIAN;
}
    
/* Description of the cuda_check_for_convergence function
* =======================================================
*
* This function checks after each iteration whether the fits are converged or not.
* It also checks whether the set maximum number of iterations is reached.
*
* Parameters:
*
* finished: An input and output vector which allows the calculation to be skipped
*           for single fits.
*
* tolerance: The tolerance value for the convergence set by user.
*
* states: An output vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred. If the maximum
*         number of iterations is reached without converging, it is set to 1. If
*         the fit converged it keeps its initial value of 0.
*
* chi_squares: An input vector of chi-square values for multiple fits. Used for the
*              convergence check.
*
* prev_chi_squares: An input vector of chi-square values for multiple fits calculated
*                   in the previous iteration. Used for the convergence check.
*
* iteration: The value of the current iteration. It is compared to the value
*            of the maximum number of iteration set by user.
*
* max_n_iterations: The maximum number of iterations set by user.
*
* // NEW
* num_v_coefs: number of coefficients to be fitted. Mainly for linear_1d, ie. polynomial fit function
*
* n_fits: The number of fits.
*
* Calling the cuda_check_for_convergence function
* ===============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_check_for_convergence<<< blocks, threads >>>(
*       finished,
*       tolerance,
*       states,
*       chi_squares,
*       prev_chi_squares,
*       iteration,
*       max_n_iterations,
*       n_fits);
*
*/

__global__ void cuda_check_for_convergence(
    int * finished,
    REAL const tolerance,
    int * states,
    REAL const * chi_squares,
    REAL const * prev_chi_squares,
    int const iteration,
    int const max_n_iterations,
    int const n_fits)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (finished[fit_index])
    {
        return;
    }

    int const fit_found
        = abs(chi_squares[fit_index] - prev_chi_squares[fit_index])
        < tolerance * max(1., chi_squares[fit_index]);

    int const max_n_iterations_reached = iteration == max_n_iterations - 1;

    if (fit_found)
    {
        finished[fit_index] = 1;
    }
    else if (max_n_iterations_reached)
    {
        states[fit_index] = MAX_ITERATION;
    }
}

/* Description of the cuda_evaluate_iteration function
* ====================================================
*
* This function evaluates the current iteration.
*   - It marks a fit as finished if a problem occured.
*   - It saves the needed number of iterations if a fit finished.
*   - It checks if all fits finished
*
* Parameters:
*
* all_finished: An output flag, that indicates whether all fits finished.
*
* n_iterations: An output vector of needed iterations for each fit.
*
* finished: An input and output vector which allows the evaluation to be skipped
*           for single fits
*
* iteration: The values of the current iteration.
*
* states: An input vector of values which indicate whether the fitting process
*         was carreid out correctly or which problem occurred.
*
* n_fits: The number of fits.
*
* Calling the cuda_evaluate_iteration function
* ============================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_evaluate_iteration<<< blocks, threads >>>(
*       all_finished,
*       n_iterations,
*       finished,
*       iteration,
*       states,
*       n_fits);
*
*/

__global__ void cuda_evaluate_iteration(
    int * all_finished,
    int * n_iterations,
    int * finished,
    int const iteration,
    int const * states,
    int const n_fits)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (states[fit_index] != CONVERGED)
    {
        finished[fit_index] = 1;
    }

    if (finished[fit_index] && n_iterations[fit_index] == 0)
    {
        n_iterations[fit_index] = iteration + 1;
    }

    if (!finished[fit_index])
    {
        *all_finished = 0;
    }
}

/* Description of the cuda_prepare_next_iteration function
* ========================================================
*
* This function prepares the next iteration. It either updates previous
* chi-square values or sets currently calculated chi-square values and
* parameters to values calculated by the previous iteration. This function also
* updates lambda values.
*
* Parameters:
*
* lambdas: An output vector of values which control the step width by modifying
*          the diagonal elements of the hessian matrices.
*
* chi_squares: An input and output vector of chi-square values for multiple fits.
*
* prev_chi_squares: An input and output vector of chi-square values for multiple
*                   fits calculated in the previous iteration.
*
* parameters: An output vector of concatenated sets of model parameters.
*
* prev_parameters: An input vector of concatenated sets of model parameters
*                  calculated in the previous iteration.
*
* n_fits: The number of fits.
*
* n_parameters: The number of fitting curve parameters.
*
* Calling the cuda_prepare_next_iteration function
* ================================================
*
* When calling the function, the blocks and threads must be set up correctly,
* as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   int const example_value = 256;
*
*   threads.x = min(n_fits, example_value);
*   blocks.x = int(ceil(REAL(n_fits) / REAL(threads.x)));
*
*   cuda_prepare_next_iteration<<< blocks, threads >>>(
*       lambdas,
*       chi_squares,
*       prev_chi_squares,
*       parameters,
*       prev_parameters,
*       n_fits,
*       n_parameters);
*
*/

__global__ void cuda_prepare_next_iteration(
    REAL * lambdas,
    REAL * chi_squares,
    REAL * prev_chi_squares,
    REAL * parameters,
    REAL const * prev_parameters,
    int const n_fits,
    int const n_parameters)
{
    int const fit_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (fit_index >= n_fits)
    {
        return;
    }

    if (chi_squares[fit_index] < prev_chi_squares[fit_index])
    {
        lambdas[fit_index] *= 0.1f;
        prev_chi_squares[fit_index] = chi_squares[fit_index];
    }
    else
    {
        lambdas[fit_index] *= 10.;
        chi_squares[fit_index] = prev_chi_squares[fit_index];
        for (int iparameter = 0; iparameter < n_parameters; iparameter++)
        {
            parameters[fit_index * n_parameters + iparameter] = prev_parameters[fit_index * n_parameters + iparameter];
        }
    }
}
/* -------------------------------------------------------------------------------------------------------
* from cuda_kernel.cu END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from info.h START
------------------------------------------------------------------------------------------------------- */
class Info
{
public:
    Info();
    virtual ~Info();

    void set_fits_per_block(std::size_t const n_fits);
    void set_number_of_parameters_to_fit(int const * parameters_to_fit);
    void configure();

private:
    void get_gpu_properties();
    void set_max_chunk_size();
    void set_blocks_per_fit();

public:
    int n_parameters_;
    int n_parameters_to_fit_;

    int n_points_;
    int power_of_two_n_points_;

    std::size_t n_fits_;

    std::size_t user_info_size_;

    int max_n_iterations_;
    int num_v_coefs_; // NEW
    std::size_t max_chunk_size_;

    int n_fits_per_block_;
    int n_blocks_per_fit_;
    ModelID model_id_;
    EstimatorID estimator_id_;

    bool use_weights_;

    int max_threads_;
    int warp_size_;

    DataLocation data_location_;

private:
    std::size_t max_blocks_;
    std::size_t available_gpu_memory_;
};

int getDeviceCount();
/* -------------------------------------------------------------------------------------------------------
* from info.h END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from info.cu START
------------------------------------------------------------------------------------------------------- */
void Info::get_gpu_properties()
{
    hipDeviceProp_t devProp;
    CUDA_CHECK_STATUS(hipGetDeviceProperties(&devProp, 0));
    max_threads_ = devProp.maxThreadsPerBlock;
    max_blocks_ = devProp.maxGridSize[0];
    warp_size_ = devProp.warpSize;

    std::size_t free_bytes;
    std::size_t total_bytes;
    CUDA_CHECK_STATUS(hipMemGetInfo(&free_bytes, &total_bytes));
    available_gpu_memory_ = std::size_t(double(free_bytes) * 0.1);
    
    if (double(user_info_size_) > double(free_bytes) * 0.9)
    {
        throw std::runtime_error("maximum user info size exceeded");
    }
}

int getDeviceCount()
{
	int deviceCount;
	CUDA_CHECK_STATUS(hipGetDeviceCount(&deviceCount));
	return deviceCount;
}
/* -------------------------------------------------------------------------------------------------------
* from info.cu END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from info.cpp START
------------------------------------------------------------------------------------------------------- */
Info::Info() :
    n_parameters_(0),
    n_parameters_to_fit_(0),
    max_chunk_size_(0),
    max_n_iterations_(0),
    num_v_coefs_(0), // NEW
    n_points_(0),
    power_of_two_n_points_(0),
    n_fits_(0),
    user_info_size_(0),
    n_fits_per_block_(0),
    n_blocks_per_fit_(0),
    max_threads_(0),
    max_blocks_(0),
    warp_size_(0),
    available_gpu_memory_(0)
{
}

Info::~Info(void)
{
}

void Info::set_number_of_parameters_to_fit(int const * const parameters_to_fit)
{
    n_parameters_to_fit_ = n_parameters_;

    for (int i = 0; i < n_parameters_; i++)
    {
        if (!parameters_to_fit[i])
        {
            n_parameters_to_fit_--;
        }
    }
}

void Info::set_fits_per_block(std::size_t const current_chunk_size)
{

    n_fits_per_block_ = std::max((max_threads_ / power_of_two_n_points_), 1);

    bool is_divisible = current_chunk_size % n_fits_per_block_ == 0;

    while (!is_divisible && (n_fits_per_block_ > 1))
    {
        n_fits_per_block_ -= 1;
        is_divisible = current_chunk_size % n_fits_per_block_ == 0;
    }

}

void Info::set_blocks_per_fit()
{
    n_blocks_per_fit_ = 1;
    
    if (power_of_two_n_points_ > max_threads_)
    {
        bool enough_threads = false;
        do
        {
            n_blocks_per_fit_ *= 2;
            enough_threads = power_of_two_n_points_ / n_blocks_per_fit_ < max_threads_;
        } while (!enough_threads);
    }
}

void Info::set_max_chunk_size()
{
    int one_fit_memory
        = sizeof(REAL)
        *(1 * n_points_                                     // values
        + 1 * n_parameters_                                 // prev_parameters
        + 1 * n_parameters_to_fit_                          // gradient
        + 1 * n_parameters_to_fit_ * n_parameters_to_fit_   // hessian
        + 2 * n_parameters_to_fit_                          // delta, scaling_vector
        + 1 * n_points_*n_parameters_                       // derivatives
        + 2)                                                // prev_chi_square, lambda,
                                                            
        + sizeof(int)
        *(1 * n_parameters_to_fit_                          // indices of fitted parameters
        + 3);                                               // finished, iteration failed flag,
                                                            // solution info
    if (n_blocks_per_fit_ > 1)
    {
        one_fit_memory
            += sizeof(REAL)
             * n_parameters_to_fit_ * n_blocks_per_fit_;    // subtotals
    }

    if (data_location_ == HOST)
    {
        one_fit_memory += sizeof(REAL) * n_points_;        // data
        one_fit_memory += sizeof(REAL) * n_parameters_;    // parameters
        one_fit_memory += sizeof(REAL);                    // chi-square
        one_fit_memory += sizeof(int) * 2;                  // state, number of iterations
        if (use_weights_)
            one_fit_memory += sizeof(REAL) * n_points_;    // weights
    }

#ifdef USE_CUBLAS
    one_fit_memory
        += sizeof(REAL)
        *(2                                                 // pointer to decomposed hessian, pointer to delta
        + 1 * n_parameters_to_fit_ * n_parameters_to_fit_)  // decomposed hessian
        + sizeof(int)
        * (1 * n_parameters_to_fit_);                       // pivot vector
#endif // USE_CUBLAS
    
    std::size_t tmp_chunk_size = available_gpu_memory_ / one_fit_memory;
    
    if (tmp_chunk_size == 0)
    {
        throw std::runtime_error("not enough free GPU memory available");
    }

    tmp_chunk_size = (std::min)(tmp_chunk_size, max_blocks_ / n_blocks_per_fit_);

    std::size_t const highest_factor = n_points_ * n_parameters_;

    std::size_t const highest_size_t_value = std::numeric_limits< std::size_t >::max();

    if (tmp_chunk_size > highest_size_t_value / highest_factor)
    {
        tmp_chunk_size = highest_size_t_value / highest_factor;
    }

    max_chunk_size_ = tmp_chunk_size;

    int i = 1;
    int const divisor = 10;
    while (tmp_chunk_size > divisor)
    {
        i *= divisor;
        tmp_chunk_size /= divisor;
    }
    max_chunk_size_ = (max_chunk_size_ / i) * i;
    max_chunk_size_ = std::min(max_chunk_size_, n_fits_);
}


void Info::configure()
{
    power_of_two_n_points_ = 1;
    while (power_of_two_n_points_ < n_points_)
    {
        power_of_two_n_points_ *= 2;
    }
    
    // TODO NOTE: to address the 'too many resources requested' error due to complex calculate_acf1D function.
    // increasing the power_of_two_n_points_ by 4 times would reduce the number of threads required in Info::set_fits_per_block function
    // Info::set_blocks_per_fit functions function would also be updated accordingly.
    power_of_two_n_points_ *= 4;

    get_gpu_properties();
    set_blocks_per_fit();
    set_max_chunk_size();
}
/* -------------------------------------------------------------------------------------------------------
* from info.cpp END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from gpu_data.cuh START
------------------------------------------------------------------------------------------------------- */
template< typename Type >
struct Device_Array
{
    explicit Device_Array(std::size_t const size) : allocated_size_(size)
    {
        std::size_t const maximum_size = std::numeric_limits< std::size_t >::max();
        std::size_t const type_size = sizeof(Type);
        if (size <= maximum_size / type_size)
        {
            hipError_t const status = hipMalloc(&data_, size * type_size);
            if (status == hipSuccess)
            {
                return;
            }
            else
            {
                throw std::runtime_error(hipGetErrorString(status));
            }
        }
        else
        {
            throw std::runtime_error("maximum array size exceeded");
        }
    }

    ~Device_Array() { if (allocated_size_ > 0) hipFree(data_); }

    operator Type * () { return static_cast<Type *>(data_); }
    operator Type const * () const { return static_cast<Type *>(data_); }

    Type const * data() const
    {
        return static_cast<Type *>(data_);
    }

    void assign(Type const * data)
    {
        data_ = const_cast<Type *>(data);
    }

    Type * copy(std::size_t const size, Type * const to) const
    {
        // TODO check size parameter
        std::size_t const type_size = sizeof(Type);
        hipError_t const status
            = hipMemcpy(to, data_, size * type_size, hipMemcpyDeviceToHost);
        if (status == hipSuccess)
        {
            return to + size;
        }
        else
        {
            throw std::runtime_error(hipGetErrorString(status));
        }
    }

private:
    void * data_;
    std::size_t allocated_size_;
};

class GPUData
{
public:
    GPUData(Info const & info);
    ~GPUData();

    void init
    (
        int const chuk_size,
        int const chunk_index,
        REAL const * data,
        REAL const * weights,
        REAL const * initial_parameters,
        std::vector<int> const & parameters_to_fit_indices,
        int * states,
        REAL * chi_squares,
        int * n_iterations
    );
    void init_user_info(char const * user_info);

    void read(bool * dst, int const * src);
    void set(int* arr, int const value);
    void set(REAL* arr, REAL const value, int const count);
    void copy(REAL * dst, REAL const * src, std::size_t const count);

private:

    void set(int* arr, int const value, int const count);
    void write(REAL* dst, REAL const * src, int const count);
    void write(int* dst, std::vector<int> const & src);
    void write(char* dst, char const * src, std::size_t const count);
    void point_to_data_sets();

private:
    int chunk_size_;
    Info const & info_;

public:
    int chunk_index_;

    hipblasHandle_t cublas_handle_;

    Device_Array< REAL > data_;
    Device_Array< REAL > weights_;
    Device_Array< REAL > parameters_;
    Device_Array< REAL > prev_parameters_;
    Device_Array< int > parameters_to_fit_indices_;
    Device_Array< char > user_info_;

    Device_Array< REAL > chi_squares_;
    Device_Array< REAL > prev_chi_squares_;
    Device_Array< REAL > gradients_;
    Device_Array< REAL > hessians_;
    Device_Array< REAL > deltas_;
    Device_Array< REAL > scaling_vectors_;
    Device_Array< REAL > subtotals_;

    Device_Array< REAL > values_;
    Device_Array< REAL > derivatives_;

    Device_Array< REAL > lambdas_;
    Device_Array< int > states_;
    Device_Array< int > finished_;
    Device_Array< int > iteration_failed_;
    Device_Array< int > all_finished_;
    Device_Array< int > n_iterations_;
    Device_Array< int > solution_info_;

#ifdef USE_CUBLAS
    Device_Array< REAL > decomposed_hessians_;
    Device_Array< REAL * > pointer_decomposed_hessians_;
    Device_Array< REAL * > pointer_deltas_;
    Device_Array< int > pivot_vectors_;
#endif // USE_CUBLAS
};
/* -------------------------------------------------------------------------------------------------------
* from gpu_data.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from gpu_data.cu START
------------------------------------------------------------------------------------------------------- */
GPUData::GPUData(Info const & info) :
    chunk_size_(0),
    info_(info),

    data_(
        (info_.data_location_ == HOST)
        ? info_.max_chunk_size_*info_.n_points_ : 0),
    weights_( 
        (info_.use_weights_ && info_.data_location_ == HOST)
        ? info_.n_points_ * info_.max_chunk_size_ : 0 ),
    parameters_(
        (info_.data_location_ == HOST)
        ? info_.max_chunk_size_*info_.n_parameters_ : 0 ),
    user_info_(
        (info_.data_location_ == HOST)
        ? info_.user_info_size_ : 0),

    prev_parameters_( info_.max_chunk_size_*info_.n_parameters_ ),
    parameters_to_fit_indices_( info_.n_parameters_to_fit_ ),

    chi_squares_(
        (info_.data_location_ == HOST)
        ? info_.max_chunk_size_ : 0),

    prev_chi_squares_( info_.max_chunk_size_ ),
    gradients_( info_.max_chunk_size_ * info_.n_parameters_to_fit_),
    hessians_( info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_ ),
    deltas_(info_.max_chunk_size_ * info_.n_parameters_to_fit_),
    scaling_vectors_(info_.max_chunk_size_ * info_.n_parameters_to_fit_),

    subtotals_(
        (info_.n_blocks_per_fit_ > 1)
        ? info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_blocks_per_fit_ : 0),

    values_( info_.max_chunk_size_ * info_.n_points_ ),
    derivatives_( info_.max_chunk_size_ * info_.n_points_ * info_.n_parameters_ ),

    lambdas_( info_.max_chunk_size_ ),

    states_(
        (info_.data_location_ == HOST)
        ? info_.max_chunk_size_ : 0),
    
    finished_( info_.max_chunk_size_ ),
    iteration_failed_(info_.max_chunk_size_),
    all_finished_( 1 ),

    n_iterations_(
        (info_.data_location_ == HOST)
        ? info_.max_chunk_size_ : 0),
    
    solution_info_(info_.max_chunk_size_)

#ifdef USE_CUBLAS
    ,
    decomposed_hessians_(info_.max_chunk_size_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_),
    pointer_decomposed_hessians_(info_.max_chunk_size_),
    pointer_deltas_(info_.max_chunk_size_),
    pivot_vectors_(info_.max_chunk_size_ * info_.n_parameters_to_fit_)
#endif // USE_CUBLAS
{
#ifdef USE_CUBLAS
    hipblasCreate(&cublas_handle_);
    point_to_data_sets();
#endif // USE_CUBLAS
}

GPUData::~GPUData()
{
#ifdef USE_CUBLAS
    hipblasDestroy(cublas_handle_);
#endif // USE_CUBLAS
}

void GPUData::init
(
    int const chunk_size,
    int const chunk_index,
    REAL const * const data,
    REAL const * const weights,
    REAL const * const initial_parameters,
    std::vector<int> const & parameters_to_fit_indices,
    int * states,
    REAL * chi_squares,
    int * n_iterations)
{
    chunk_size_ = chunk_size;
    chunk_index_ = chunk_index;

    if (info_.data_location_ == HOST)
    {
        write(
            data_,
            data + chunk_index_*info_.max_chunk_size_*info_.n_points_,
            chunk_size_ * info_.n_points_);
        write(
            parameters_,
            initial_parameters + chunk_index_*info_.max_chunk_size_*info_.n_parameters_,
            chunk_size_ * info_.n_parameters_);
        if (info_.use_weights_)
            write(
                weights_,
                weights + chunk_index_*info_.max_chunk_size_*info_.n_points_,
                chunk_size_ * info_.n_points_);
    }
    else if (info_.data_location_ == DEVICE)
    {
        data_.assign(
            data + chunk_index_*info_.max_chunk_size_*info_.n_points_);
        parameters_.assign(
            initial_parameters + chunk_index_*info_.max_chunk_size_*info_.n_parameters_);
        if (info_.use_weights_)
            weights_.assign(
                weights + chunk_index_*info_.max_chunk_size_*info_.n_points_);
        states_.assign(
            states + chunk_index_ * info_.max_chunk_size_);
        chi_squares_.assign(
            chi_squares + chunk_index_ * info_.max_chunk_size_);
        n_iterations_.assign(
            n_iterations + chunk_index_ * info_.max_chunk_size_);
    }

    write(parameters_to_fit_indices_, parameters_to_fit_indices);

    set(prev_chi_squares_, 0., chunk_size_);
    set(finished_, 0, chunk_size_);
    set(scaling_vectors_, 0., chunk_size_ * info_.n_parameters_to_fit_);
    set(states_, 0, chunk_size_);
    set(lambdas_, 0.001f, chunk_size_);
}

void GPUData::init_user_info(char const * const user_info)
{
    if (info_.user_info_size_ > 0)
    {
        if (info_.data_location_ == HOST)
        {
            write(user_info_, user_info, info_.user_info_size_);
        }
        else if (info_.data_location_ == DEVICE)
        {
            user_info_.assign(user_info);
        }
    }
}

void GPUData::read(bool * dst, int const * src)
{
    int int_dst = 0;
    CUDA_CHECK_STATUS(hipMemcpy(&int_dst, src, sizeof(int), hipMemcpyDeviceToHost));
    * dst = (int_dst == 1) ? true : false;
}

void GPUData::write(REAL* dst, REAL const * src, int const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(REAL), hipMemcpyHostToDevice));
}

void GPUData::write(int* dst, std::vector<int> const & src)
{
    std::size_t const size = src.size() * sizeof(int);
    CUDA_CHECK_STATUS(hipMemcpy(dst, src.data(), size, hipMemcpyHostToDevice));
}

void GPUData::write(char* dst, char const * src, std::size_t const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(char), hipMemcpyHostToDevice));
}

void GPUData::copy(REAL * dst, REAL const * src, std::size_t const count)
{
    CUDA_CHECK_STATUS(hipMemcpy(dst, src, count * sizeof(REAL), hipMemcpyDeviceToDevice));
}

__global__ void set_kernel(int* dst, int const value, int const count)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= count)
        return;

    dst[index] = value;
}

void GPUData::set(int* arr, int const value, int const count)
{
    int const tx = 256;
	int const bx = (count / tx) + 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);

    set_kernel<<< blocks, threads >>>(arr, value, count);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void GPUData::set(int* arr, int const value)
{
    int const tx = 1;
    int const bx = 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);

    set_kernel<<< blocks, threads >>>(arr, value, 1);
    CUDA_CHECK_STATUS(hipGetLastError());
}

__global__ void set_kernel(REAL* dst, REAL const value, std::size_t const count)
{
	std::size_t const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= count)
        return;

    dst[index] = value;
}

void GPUData::set(REAL* arr, REAL const value, int const count)
{
    int const tx = 256;
	int const bx = (count / tx) + 1;

    dim3  threads(tx, 1, 1);
    dim3  blocks(bx, 1, 1);
    set_kernel<<< blocks, threads >>>(arr, value, count);
    CUDA_CHECK_STATUS(hipGetLastError());
}

__global__ void cuda_point_to_data_sets(
    REAL ** pointer_to_pointers,
    REAL * pointer,
    std::size_t const n_pointers,
    std::size_t const size)
{
    std::size_t const index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n_pointers)
        return;

    int const begin = index * size;

    pointer_to_pointers[index] = pointer + begin;
}
#ifdef USE_CUBLAS

void GPUData::point_to_data_sets()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    std::size_t max_threads = 256;

    threads.x
        = static_cast<unsigned int>
          (std::min(info_.max_chunk_size_, max_threads));
    blocks.x
        = static_cast<unsigned int>
          (std::ceil(REAL(info_.max_chunk_size_) / REAL(threads.x)));

    cuda_point_to_data_sets <<< blocks, threads >>>(
        pointer_decomposed_hessians_,
        decomposed_hessians_,
        info_.max_chunk_size_,
        info_.n_parameters_to_fit_*info_.n_parameters_to_fit_);

    cuda_point_to_data_sets <<< blocks, threads >>> (
        pointer_deltas_,
        deltas_,
        info_.max_chunk_size_,
        info_.n_parameters_to_fit_);
}

#endif // USE_CUBLAS
/* -------------------------------------------------------------------------------------------------------
* from gpu_data.cu END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from cuda_gaussjordan.cuh START
------------------------------------------------------------------------------------------------------- */
/*
extern __global__ void cuda_gaussjordan(
    REAL * delta,
    REAL const * beta,
    REAL const * alpha,
    int const * skip_calculation,
    int * singular,
    std::size_t const n_equations,
    std::size_t const n_equations_pow2);
*/
/* -------------------------------------------------------------------------------------------------------
* from cuda_gaussjordan.cuh END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from cuda_gaussjordan.cu START
------------------------------------------------------------------------------------------------------- */
/* CUDA implementation of Gauss-Jordan elimination algorithm.
*  
* Gauss-Jordan elimination method
* ===============================
*
* This function solves a set of linear equations using the Gauss-Jordan elimination method.
* Considering a set of N equations with N unknowns, this can be written in matrix form as
* an NxN matrix of coefficients and a Nx1 column vector of right-hand side values.
*
* For example, consider the following problem with 3 equations and 3 unknowns (N=3):
* 
*   A x + B y + C z = MM
*   D x + E y + F z = NN
*   G x + H y + J z = PP
* 
* We can write this as follows in matrix form:
* 
*   [ A B C ] [ x ] = [ MM ]
*   [ D E F ] [ y ] = [ NN ] 
*   [ G H I ] [ z ] = [ PP ]
* 
* or, [A]*[X] = [B] where [A] is the matrix of coefficients and [B] is the vector of 
* right-hand side values.
*
* The Gauss Jordan elimiation method solves the system of equations in the following
* manner.  First, we form the augmented matrix (A|B):
*
*   [ A B C | MM ] 
*   [ D E F | NN ] 
*   [ G H I | PP ] 
*
* and then the augmented matrix is manipulated until its left side has the reduced
* row-echelon form.  That is to say that any individual row may be multiplied
* by a scalar factor, and any linear combination of rows may be added to another 
* row.  Finally, two rows may be swapped without affecting the solution.
* 
* When the manipulations are complete and the left side of the matrix has the desired
* form, the right side then corresponds to the solution of the system. 
*
*
* Description of the cuda_gaussjordan function
* ============================================
* 
* This algorithm is designed to perform many solutions of the Gauss Jordan elimination
* method in parallel.  One limitation of the algorithm implemented here is that for
* each solution the number of equations and unknowns (N) must be identical.  
*
* Parameters:
* 
* alpha: Coefficients matrices.  The matrix of coefficients for a single solution is 
*        a vector of NxN, where N is the number of equations.  This array stores the 
*        coefficients for the entire set of M input problems, concatenated end to end, 
*        and hence the total size of the array is MxNxN.  
*
* beta: Vector of right hand side values, concatenated together for all input problems. 
*       For a set of M inputs, the size of the vector is MxN.  Upon completion, this 
*       vector contains the results vector X for each solution.
*
* skip_calculation: An input vector which allows the calculation to be skipped for
*                   a particular solution.  For a set of M inputs, the size of this
*                   vector is M. 
*
* singular: An output vector used to report whether a given solution is singular.  For
*           a set of M inputs, this vector has size M.  Memory needs to be allocated
*           by the calling the function.
*
* n_equations: The number of equations and unknowns for a single solution.  This is
*              equal to the size N.
*
* n_equations_pow2: The next highest power of 2 greater than n_equations.
*
*
* Calling the cuda_gaussjordan function
* =====================================
*
* When calling the function, the blocks and threads must be set up correctly, as well
* as the shared memory space, as shown in the following example code.
*
*   dim3  threads(1, 1, 1);
*   dim3  blocks(1, 1, 1);
*
*   threads.x = n_equations + 1;
*   threads.y = n_equations;
*   blocks.x = n_solutions;
*   blocks.y = 1;
*
*   int const shared_size = sizeof(REAL) * 
*       ( (threads.x * threads.y) + n_parameters_pow2 + n_parameters_pow2 );
*
*   int * singular;
*   CUDA_CHECK_STATUS(hipMalloc((void**)&singular, n_solutions * sizeof(int)));
*
*   cuda_gaussjordan<<< blocks, threads, shared_size >>>(
*       alpha,
*       beta,
*       skip_calculation,
*       singular,
*       n_equations,
*       n_equations_pow2);
*
*/

__global__ void cuda_gaussjordan(
    REAL * delta,
    REAL const * beta,
    REAL const * alpha,
    int const * skip_calculation,
    int * singular,
    std::size_t const n_equations,
    std::size_t const n_equations_pow2)
{
    extern __shared__ REAL extern_array[];     //shared memory between threads of a single block, 
    //used for storing the calculation_matrix, the 
    //abs_row vector, and the abs_row_index vector

    // In this routine we will store the augmented matrix (A|B), referred to here
    // as the calculation matrix in a shared memory space which is visible to all
    // threads within a block.  Also stored in shared memory are two vectors which 
    // are used to find the largest element in each row (the pivot).  These vectors 
    // are called abs_row and abs_row_index.
    //
    // Sizes of data stored in shared memory:
    //
    //      calculation_matrix: n_equations * (n_equations+1)
    //      abs_row:            n_equations_pow2
    //      abs_row_index:      n_equations_pow2
    //  
    // Note that each thread represents an element of the augmented matrix, with
    // the column and row indicated by the x and y index of the thread.  Each 
    // solution is calculated within one block, and the solution index is the 
    // block index x value.

    int const col_index = threadIdx.x;                  //column index in the calculation_matrix
    int const row_index = threadIdx.y;                  //row index in the calculation_matrix
    int const solution_index = blockIdx.x;

    int const n_col = blockDim.x;                       //number of columns in calculation matrix (=threads.x)
    int const n_row = blockDim.y;                       //number of rows in calculation matrix (=threads.y)
    int const alpha_size = blockDim.y * blockDim.y;     //number of entries in alpha matrix for one solution (NxN)

    if (skip_calculation[solution_index])
        return;

    REAL p;                                            //local variable used in pivot calculation

    REAL * calculation_matrix = extern_array;                          //point to the shared memory

    REAL * abs_row = extern_array + n_equations * (n_equations + 1);     //abs_row is located after the calculation_matrix
    //within the shared memory

    int * abs_row_index = (int *)(abs_row + n_equations_pow2);            //abs_row_index is located after abs_row
    //
    //note that although the shared memory is defined as
    //REAL, we are storing data of type int in this
    //part of the shared memory

    //initialize the singular vector
    if (col_index == 0 && row_index == 0)
    {
        singular[solution_index] = 0;
    }

    //initialize abs_row and abs_row_index, using only the threads on the diagonal
    if (col_index == row_index)
    {
        abs_row[col_index + (n_equations_pow2 - n_equations)] = 0.0;
        abs_row_index[col_index + (n_equations_pow2 - n_equations)] = col_index + (n_equations_pow2 - n_equations);
    }

    //initialize the calculation_matrix (alpha and beta, concatenated, for one solution)
    if (col_index != n_equations)
        calculation_matrix[row_index*n_col + col_index] = alpha[solution_index * alpha_size + row_index * n_equations + col_index];
    else
        calculation_matrix[row_index*n_col + col_index] = beta[solution_index * n_equations + row_index];

    //wait for thread synchronization

    __syncthreads();

    //start of main outer loop over the rows of the calculation matrix

    for (int current_row = 0; current_row < n_equations; current_row++)
    {

        // work in only one row, skipping the last column
        if (row_index == current_row && col_index != n_equations)
        {

            //save the absolute values of the current row
            abs_row[col_index] = abs(calculation_matrix[row_index * n_col + col_index]);

            //save the column indices
            abs_row_index[col_index] = col_index;

            __threadfence();

            //find the largest absolute value in the current row and write its index in abs_row_index[0]
            for (int n = 2; n <= n_equations_pow2; n = n * 2)
            {
                if (col_index < (n_equations_pow2 / n))
                {
                    if (abs_row[abs_row_index[col_index]] < abs_row[abs_row_index[col_index + (n_equations_pow2 / n)]])
                    {
                        abs_row_index[col_index] = abs_row_index[col_index + (n_equations_pow2 / n)];
                    }
                }
            }
        }

        __syncthreads();

        //singularity check - if all values in the row are zero, no solution exists
        if (row_index == current_row && col_index != n_equations)
        {
            if (abs_row[abs_row_index[0]] == 0.0)
            {
                singular[solution_index] = 1;
            }
        }

        //devide the row by the biggest value in the row
        if (row_index == current_row)
        {
            calculation_matrix[row_index * n_col + col_index]
                = calculation_matrix[row_index * n_col + col_index] / calculation_matrix[row_index * n_col + abs_row_index[0]];
        }

        __syncthreads();

        //The value of the largest element of the current row was found, and then current
        //row was divided by this value such that the largest value of the current row 
        //is equal to one.  
        //
        //Next, the matrix is manipulated to reduce to zero all other entries in the column 
        //in which the largest value was found.   To do this, the values in the current row
        //are scaled appropriately and substracted from the other rows of the matrix. 
        //
        //For each element of the matrix that is not in the current row, calculate the value
        //to be subtracted and let each thread store this value in the scalar variable p.

        p = calculation_matrix[current_row * n_col + col_index] * calculation_matrix[row_index * n_col + abs_row_index[0]];
        __syncthreads();

        if (row_index != current_row)
        {
            calculation_matrix[row_index * n_col + col_index] = calculation_matrix[row_index * n_col + col_index] - p;
        }
        __syncthreads();

    }

    //At this point, if the solution exists, the calculation matrix has been reduced to the 
    //identity matrix on the left side, and the solution vector on the right side.  However
    //we have not swapped rows during the procedure, so the identity matrix is out of order.
    //
    //For example, starting with the following augmented matrix as input:
    //
    //  [  3  2 -4 |  4 ]
    //  [  2  3  3 | 15 ]
    //  [  5 -3  1 | 14 ]
    //
    //we will obtain:
    //
    //  [  0  0  1 |  2 ]
    //  [  0  1  0 |  1 ]
    //  [  1  0  0 |  3 ]
    //
    //Which needs to be re-arranged to obtain the correct solution vector.  In the final
    //step, each thread checks to see if its value equals 1, and if so it assigns the value
    //in its rightmost column to the appropriate entry in the beta vector.  The solution is
    //stored in beta upon completetion.

    if (col_index != n_equations && calculation_matrix[row_index * n_col + col_index] == 1)
        delta[n_row * solution_index + col_index] = calculation_matrix[row_index * n_col + n_equations];

    __syncthreads();
}
/* -------------------------------------------------------------------------------------------------------
* from cuda_gaussjordan.cu END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from lm_fit.h START
------------------------------------------------------------------------------------------------------- */
/*
class LMFitCUDA;
*/

class LMFit
{
public:
    LMFit
    (
        REAL const * data,
        REAL const * weights,
        Info & info,
        REAL const * initial_parameters,
        int const * parameters_to_fit,
        char * user_info,
        REAL * output_parameters,
        int * output_states,
        REAL * output_chi_squares,
        int * output_n_iterations
    ) ;

    virtual ~LMFit();

    void run(REAL const tolerance);

private:
    void set_parameters_to_fit_indices();
    void get_results(GPUData const & gpu_data, int const n_fits);

    REAL const * const data_ ;
    REAL const * const weights_ ;
    REAL const * const initial_parameters_ ;
    int const * const parameters_to_fit_;
    char const * const user_info_;

    REAL * output_parameters_ ;
    int * output_states_ ;
    REAL * output_chi_squares_ ;
    int * output_n_iterations_ ;

    int ichunk_;
    int chunk_size_;
    std::size_t n_fits_left_;

    Info & info_;

    std::vector<int> parameters_to_fit_indices_;
};

class LMFitCUDA
{
public:
    LMFitCUDA(
        REAL const tolerance,
        Info const & info,
        GPUData & gpu_data,
        int const n_fits);

    virtual ~LMFitCUDA();

    void run();

private:
    void calc_curve_values();
    void calc_chi_squares();
    void calc_gradients();
    void calc_hessians();
    void evaluate_iteration(int const iteration);
    void scale_hessians();
#ifdef USE_CUBLAS
    void solve_equation_systems_lup();
#else
    void solve_equation_systems_gj();
#endif
    void update_states();
    void update_parameters();

public:

private:
    Info const & info_;
    GPUData & gpu_data_;
    int const n_fits_;

    bool all_finished_;

    REAL tolerance_;
};
/* -------------------------------------------------------------------------------------------------------
* from lm_fit.h END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from lm_fit.cpp START
------------------------------------------------------------------------------------------------------- */
LMFit::LMFit
(
    REAL const * const data,
    REAL const * const weights,
    Info & info,
    REAL const * const initial_parameters,
    int const * const parameters_to_fit,
    char * const user_info,
    REAL * output_parameters,
    int * output_states,
    REAL * output_chi_squares,
    int * output_n_iterations
) :
    data_( data ),
    weights_( weights ),
    initial_parameters_( initial_parameters ),
    parameters_to_fit_( parameters_to_fit ),
    user_info_( user_info ),
    output_parameters_( output_parameters ),
    output_states_( output_states ),
    output_chi_squares_( output_chi_squares ),
    output_n_iterations_( output_n_iterations ),
    info_(info),
    chunk_size_(0),
    ichunk_(0),
    n_fits_left_(info.n_fits_),
    parameters_to_fit_indices_(0)
{}

LMFit::~LMFit()
{}

void LMFit::set_parameters_to_fit_indices()
{
    int const n_parameters_to_fit = info_.n_parameters_;
    for (int i = 0; i < n_parameters_to_fit; i++)
    {
        if (parameters_to_fit_[i])
        {
            parameters_to_fit_indices_.push_back(i);
        }
    }
}

void LMFit::get_results(GPUData const & gpu_data, int const n_fits)
{
    if (info_.data_location_ == HOST)
    {
        output_parameters_
            = gpu_data.parameters_.copy(n_fits*info_.n_parameters_, output_parameters_);
        output_states_
            = gpu_data.states_.copy(n_fits, output_states_);
        output_chi_squares_
            = gpu_data.chi_squares_.copy(n_fits, output_chi_squares_);
        output_n_iterations_
            = gpu_data.n_iterations_.copy(n_fits, output_n_iterations_);
    }
}

void LMFit::run(REAL const tolerance)
{
    set_parameters_to_fit_indices();

    GPUData gpu_data(info_);
    gpu_data.init_user_info(user_info_);

    // loop over data chunks
    while (n_fits_left_ > 0)
    {
        chunk_size_ = int((std::min)(n_fits_left_, info_.max_chunk_size_));

        info_.set_fits_per_block(chunk_size_);

        gpu_data.init(
            chunk_size_,
            ichunk_,
            data_,
            weights_,
            initial_parameters_,
            parameters_to_fit_indices_,
            output_states_,
            output_chi_squares_,
            output_n_iterations_);

        LMFitCUDA lmfit_cuda(
            tolerance,
            info_,
            gpu_data,
            chunk_size_);

        lmfit_cuda.run();

        get_results(gpu_data, chunk_size_);

        n_fits_left_ -= chunk_size_;
        ichunk_++;

    }
}
/* -------------------------------------------------------------------------------------------------------
* from lm_fit.cpp END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from lm_fit_cuda.cu START
------------------------------------------------------------------------------------------------------- */
#ifdef USE_CUBLAS
    void LMFitCUDA::solve_equation_systems_lup()
    {
        dim3  threads(1, 1, 1);
        dim3  blocks(1, 1, 1);

        // initialize components of equation systems
        gpu_data_.copy(gpu_data_.decomposed_hessians_, gpu_data_.hessians_, n_fits_ * info_.n_parameters_to_fit_ * info_.n_parameters_to_fit_);

        // decompose hessians
        hipblasStatus_t lu_status_decopmposition = DECOMPOSE_LUP(
            gpu_data_.cublas_handle_,
            info_.n_parameters_to_fit_,
            gpu_data_.pointer_decomposed_hessians_,
            info_.n_parameters_to_fit_,
            gpu_data_.pivot_vectors_,
            gpu_data_.solution_info_,
            n_fits_);

        // initialize deltas with values of gradients
        gpu_data_.copy(gpu_data_.deltas_, gpu_data_.gradients_, n_fits_ * info_.n_parameters_to_fit_);

        // TODO: check solution_info
        int solution_info;
    
        // solve equation systems
        hipblasStatus_t lu_status_solution
            = SOLVE_LUP(
            gpu_data_.cublas_handle_,
            HIPBLAS_OP_N,
            info_.n_parameters_to_fit_,
            1,
            (REAL const **)(gpu_data_.pointer_decomposed_hessians_.data()),
            info_.n_parameters_to_fit_,
            gpu_data_.pivot_vectors_,
            gpu_data_.pointer_deltas_,
            info_.n_parameters_to_fit_,
            &solution_info,
            n_fits_);
    }
#else
    void LMFitCUDA::solve_equation_systems_gj()
    {
        dim3  threads(1, 1, 1);
        dim3  blocks(1, 1, 1);

        int n_parameters_pow2 = 1;

        while (n_parameters_pow2 < info_.n_parameters_to_fit_)
        {
            n_parameters_pow2 *= 2;
        }

        //set up to run the Gauss Jordan elimination
        int const n_equations = info_.n_parameters_to_fit_;
        int const n_solutions = n_fits_;

        threads.x = n_equations + 1;
        threads.y = n_equations;
        blocks.x = n_solutions;

        //set the size of the shared memory area for each block
        int const shared_size = sizeof(REAL) * ((threads.x * threads.y) + n_parameters_pow2 + n_parameters_pow2);

        //run the Gauss Jordan elimination
        cuda_gaussjordan <<< blocks, threads, shared_size >>>(
            gpu_data_.deltas_,
            gpu_data_.gradients_,
            gpu_data_.hessians_,
            gpu_data_.finished_,
            gpu_data_.solution_info_,
            info_.n_parameters_to_fit_,
            n_parameters_pow2);
        CUDA_CHECK_STATUS(hipGetLastError());
    }
#endif

void LMFitCUDA::update_states()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    //set up to update the lm_state_gpu_ variable with the Gauss Jordan results
    threads.x = std::min(n_fits_, 256);
    blocks.x = int(std::ceil(REAL(n_fits_) / REAL(threads.x)));

    //update the gpu_data_.states_ variable
    cuda_update_state_after_solving <<< blocks, threads >>>(
        n_fits_,
        gpu_data_.solution_info_,
        gpu_data_.finished_,
        gpu_data_.states_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::scale_hessians()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.n_parameters_to_fit_*info_.n_fits_per_block_;
    blocks.x = n_fits_ / info_.n_fits_per_block_;

    cuda_modify_step_widths <<< blocks, threads >>>(
        gpu_data_.hessians_,
        gpu_data_.lambdas_,
        gpu_data_.scaling_vectors_,
        info_.n_parameters_to_fit_,
        gpu_data_.iteration_failed_,
        gpu_data_.finished_,
        info_.n_fits_per_block_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::update_parameters()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.n_parameters_*info_.n_fits_per_block_;
    blocks.x = n_fits_ / info_.n_fits_per_block_;

    cuda_update_parameters <<< blocks, threads >>>(
        gpu_data_.parameters_,
        gpu_data_.prev_parameters_,
        gpu_data_.deltas_,
        info_.n_parameters_to_fit_,
        gpu_data_.parameters_to_fit_indices_,
        gpu_data_.finished_,
        info_.n_fits_per_block_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::calc_curve_values()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.n_points_ * info_.n_fits_per_block_ / info_.n_blocks_per_fit_;

    if (info_.n_blocks_per_fit_ > 1)
        threads.x += info_.n_points_ % threads.x;

    threads.x = threads.x;
    blocks.x = n_fits_ / info_.n_fits_per_block_ * info_.n_blocks_per_fit_;

    cuda_calc_curve_values <<< blocks, threads >>>(
        gpu_data_.parameters_,
        n_fits_,
        info_.n_points_,
        info_.n_parameters_,
        gpu_data_.finished_,
        gpu_data_.values_,
        gpu_data_.derivatives_,
        info_.n_fits_per_block_,
        info_.n_blocks_per_fit_,
        info_.model_id_,
        gpu_data_.chunk_index_,
        gpu_data_.user_info_,
        info_.user_info_size_,
        info_.num_v_coefs_); // NEW
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::calc_chi_squares()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.power_of_two_n_points_ * info_.n_fits_per_block_ / info_.n_blocks_per_fit_;
    blocks.x = n_fits_ / info_.n_fits_per_block_ * info_.n_blocks_per_fit_;

    int const shared_size = sizeof(REAL) * threads.x;

    REAL * chi_squares = 
        info_.n_blocks_per_fit_ > 1 ? gpu_data_.subtotals_ : gpu_data_.chi_squares_;

    cuda_calculate_chi_squares <<< blocks, threads, shared_size >>>(
        chi_squares,
        gpu_data_.states_,
        gpu_data_.data_,
        gpu_data_.values_,
        gpu_data_.weights_,
        info_.n_points_,
        n_fits_,
        info_.estimator_id_,
        gpu_data_.finished_,
        info_.n_fits_per_block_,
        gpu_data_.user_info_,
        info_.user_info_size_);
    CUDA_CHECK_STATUS(hipGetLastError());

    threads.x = std::min(n_fits_, 256);
    blocks.x = int(std::ceil(REAL(n_fits_) / REAL(threads.x)));

    if (info_.n_blocks_per_fit_ > 1)
    {
        cuda_sum_chi_square_subtotals <<< blocks, threads >>> (
            gpu_data_.chi_squares_,
            gpu_data_.subtotals_,
            info_.n_blocks_per_fit_,
            n_fits_,
            gpu_data_.finished_);
        CUDA_CHECK_STATUS(hipGetLastError());
    }

    cuda_check_fit_improvement <<< blocks, threads >>>(
        gpu_data_.iteration_failed_,
        gpu_data_.chi_squares_,
        gpu_data_.prev_chi_squares_,
        n_fits_,
        gpu_data_.finished_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::calc_gradients()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = info_.power_of_two_n_points_ * info_.n_fits_per_block_ / info_.n_blocks_per_fit_;
    blocks.x = n_fits_ / info_.n_fits_per_block_ * info_.n_blocks_per_fit_;

    int const shared_size = sizeof(REAL) * threads.x;

    REAL * gradients
        = info_.n_blocks_per_fit_ > 1 ? gpu_data_.subtotals_ : gpu_data_.gradients_;

    cuda_calculate_gradients <<< blocks, threads, shared_size >>>(
        gradients,
        gpu_data_.data_,
        gpu_data_.values_,
        gpu_data_.derivatives_,
        gpu_data_.weights_,
        info_.n_points_,
        n_fits_,
        info_.n_parameters_,
        info_.n_parameters_to_fit_,
        gpu_data_.parameters_to_fit_indices_,
        info_.estimator_id_,
        gpu_data_.finished_,
        gpu_data_.iteration_failed_,
        info_.n_fits_per_block_,
        gpu_data_.user_info_,
        info_.user_info_size_);
    CUDA_CHECK_STATUS(hipGetLastError());

    if (info_.n_blocks_per_fit_ > 1)
    {
        int const gradients_size = n_fits_ * info_.n_parameters_to_fit_;
        threads.x = std::min(gradients_size, 256);
        blocks.x = int(std::ceil(REAL(gradients_size) / REAL(threads.x)));

        cuda_sum_gradient_subtotals <<< blocks, threads >>> (
            gpu_data_.gradients_,
            gpu_data_.subtotals_,
            info_.n_blocks_per_fit_,
            n_fits_,
            info_.n_parameters_to_fit_,
            gpu_data_.iteration_failed_,
            gpu_data_.finished_);
        CUDA_CHECK_STATUS(hipGetLastError());
    }
}

void LMFitCUDA::calc_hessians()
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    int const n_unique_values
        = info_.n_parameters_to_fit_ * (info_.n_parameters_to_fit_ + 1) / 2;

    int n_hessians_per_block = 1;

    if (info_.n_parameters_to_fit_)
    {
        while ((n_hessians_per_block + 1) * n_unique_values < info_.warp_size_)
        {
            n_hessians_per_block++;
        }
    }

    int const temp_threads_x = n_unique_values * n_hessians_per_block;

    threads.x = std::min(temp_threads_x, info_.max_threads_);
    
    blocks.y
        = temp_threads_x / info_.max_threads_ 
        + int((temp_threads_x % info_.max_threads_) > 0);
    
    blocks.x
        = n_fits_ / n_hessians_per_block
        + int((n_fits_ % n_hessians_per_block) > 0);

    cuda_calculate_hessians <<< blocks, threads >>>(
        gpu_data_.hessians_,
        gpu_data_.data_,
        gpu_data_.values_,
        gpu_data_.derivatives_,
        gpu_data_.weights_,
        n_fits_,
        info_.n_points_,
        info_.n_parameters_,
        info_.n_parameters_to_fit_,
        gpu_data_.parameters_to_fit_indices_,
        info_.estimator_id_,
        gpu_data_.iteration_failed_,
        gpu_data_.finished_,
        gpu_data_.user_info_,
        info_.user_info_size_);
    CUDA_CHECK_STATUS(hipGetLastError());
}

void LMFitCUDA::evaluate_iteration(int const iteration)
{
    dim3  threads(1, 1, 1);
    dim3  blocks(1, 1, 1);

    threads.x = std::min(n_fits_, 256);
    blocks.x = int(std::ceil(REAL(n_fits_) / REAL(threads.x)));

    cuda_check_for_convergence<<< blocks, threads >>>(
        gpu_data_.finished_,
        tolerance_,
        gpu_data_.states_,
        gpu_data_.chi_squares_,
        gpu_data_.prev_chi_squares_,
        iteration,
        info_.max_n_iterations_,
        n_fits_);
    CUDA_CHECK_STATUS(hipGetLastError());

    gpu_data_.set(gpu_data_.all_finished_, 1);

    cuda_evaluate_iteration<<< blocks, threads >>>(
        gpu_data_.all_finished_,
        gpu_data_.n_iterations_,
        gpu_data_.finished_,
        iteration,
        gpu_data_.states_,
        n_fits_);
    CUDA_CHECK_STATUS(hipGetLastError());

    gpu_data_.read(&all_finished_, gpu_data_.all_finished_);

    cuda_prepare_next_iteration<<< blocks, threads >>>(
        gpu_data_.lambdas_,
        gpu_data_.chi_squares_,
        gpu_data_.prev_chi_squares_,
        gpu_data_.parameters_,
        gpu_data_.prev_parameters_,
        n_fits_,
        info_.n_parameters_);
    CUDA_CHECK_STATUS(hipGetLastError());
}
/* -------------------------------------------------------------------------------------------------------
* from lm_fit_cuda.cu END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from lm_fit_cuda.cpp START
------------------------------------------------------------------------------------------------------- */
LMFitCUDA::LMFitCUDA(
    REAL const tolerance,
    Info const & info,
    GPUData & gpu_data,
    int const n_fits
    ) :
    info_(info),
    gpu_data_(gpu_data),
    n_fits_(n_fits),
    all_finished_(false),
    tolerance_(tolerance)
{
}

LMFitCUDA::~LMFitCUDA()
{
}

void LMFitCUDA::run()
{
    // initialize the chi-square values
    calc_curve_values();
    calc_chi_squares();

    if (info_.n_parameters_to_fit_ == 0)
        return;

    calc_gradients();
    calc_hessians();

    gpu_data_.copy(
        gpu_data_.prev_chi_squares_,
        gpu_data_.chi_squares_,
        n_fits_);

    // loop over the fit iterations
    for (int iteration = 0; !all_finished_; iteration++)
    {
        // modify step width
        // LUP decomposition
        // update fitting parameters
        scale_hessians();
        SOLVE_EQUATION_SYSTEMS();
        update_states();
        update_parameters();

        // calculate fitting curve values and its derivatives
        // calculate chi-squares, gradients and hessians
	calc_curve_values();
        calc_chi_squares();
        calc_gradients();
        calc_hessians();

        // check which fits have converged
        // flag finished fits
        // check whether all fits finished
        // save the number of needed iterations by each fitting process
        // check whether chi-squares are increasing or decreasing
        // update chi-squares, curve parameters and lambdas
        evaluate_iteration(iteration);
    }
}
/* -------------------------------------------------------------------------------------------------------
* from lm_fit_cuda.cpp END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from interface.h START
------------------------------------------------------------------------------------------------------- */
static_assert( sizeof( int ) == 4, "32 bit 'int' type required" ) ;

class FitInterface
{
public:
    FitInterface
    (
        REAL const * data,
        REAL const * weights,
        std::size_t n_fits,
        int n_points,
        REAL tolerance,
        int max_n_iterations,
        int num_v_coefs,
        EstimatorID estimator_id,
        REAL const * initial_parameters,
        int * parameters_to_fit,
        char * user_info,
        std::size_t user_info_size,
        REAL * output_parameters,
        int * output_states,
        REAL * output_chi_squares,
        int * output_n_iterations,
        DataLocation data_location
    ) ;
    
    virtual ~FitInterface();
    void fit(ModelID const model_id);

private:
    void check_sizes();
    void configure_info(Info & info, ModelID const model_id);

public:

private:
    //input
    REAL const * const data_ ;
    REAL const * const weights_;
    REAL const * const initial_parameters_;
    int const * const parameters_to_fit_;
    char * const user_info_;
    int n_parameters_;

    std::size_t const n_fits_;
    int const n_points_;
    REAL const  tolerance_;
    int const max_n_iterations_;
    int const num_v_coefs_; // NEW
    EstimatorID estimator_id_;
    std::size_t const user_info_size_;

    DataLocation data_location_;

    //output
    REAL * output_parameters_;
    int * output_states_;
    REAL * output_chi_squares_;
    int * output_n_iterations_;
};
/* -------------------------------------------------------------------------------------------------------
* from interface.h END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from interface.cpp START
------------------------------------------------------------------------------------------------------- */
FitInterface::FitInterface
(
    REAL const * data,
    REAL const * weights,
    std::size_t n_fits,
    int n_points,
    REAL tolerance,
    int max_n_iterations,
    int num_v_coefs, // NEW
    EstimatorID estimator_id,
    REAL const * initial_parameters,
    int * parameters_to_fit,
    char * user_info,
    std::size_t user_info_size,
    REAL * output_parameters,
    int * output_states,
    REAL * output_chi_squares,
    int * output_n_iterations,
    DataLocation data_location
) :
    data_( data ),
    weights_( weights ),
    initial_parameters_( initial_parameters ),
    parameters_to_fit_( parameters_to_fit ),
    user_info_( user_info ),
    n_fits_(n_fits),
    n_points_(n_points),
    tolerance_(tolerance),
    max_n_iterations_(max_n_iterations),
    num_v_coefs_(num_v_coefs), // NEW
    estimator_id_(estimator_id),
    user_info_size_(user_info_size),
    output_parameters_( output_parameters ),
    output_states_(output_states),
    output_chi_squares_(output_chi_squares),
    output_n_iterations_(output_n_iterations),
    n_parameters_(0),
    data_location_(data_location)
{}

FitInterface::~FitInterface()
{}

void FitInterface::check_sizes()
{
    std::size_t maximum_size = std::numeric_limits< std::size_t >::max();
    
    if (n_fits_ > maximum_size / n_points_ / sizeof(REAL))
    {
        throw std::runtime_error("maximum absolute number of data points exceeded");
    }
    
    if (n_fits_ > maximum_size / n_parameters_ / sizeof(REAL))
    {
        throw std::runtime_error("maximum number of fits and/or parameters exceeded");
    }
}

void FitInterface::configure_info(Info & info, ModelID const model_id)
{
    info.model_id_ = model_id;
    info.n_fits_ = n_fits_;
    info.n_points_ = n_points_;
    info.max_n_iterations_ = max_n_iterations_;
    info.num_v_coefs_ = num_v_coefs_; // NEW
    info.estimator_id_ = estimator_id_;
    info.user_info_size_ = user_info_size_;
    info.n_parameters_ = n_parameters_;
    info.use_weights_ = weights_ ? true : false;
    info.data_location_ = data_location_;

    info.set_number_of_parameters_to_fit(parameters_to_fit_);
    info.configure();
}

void FitInterface::fit(ModelID const model_id)
{
    int n_dimensions = 0;
    configure_model(model_id, n_parameters_, n_dimensions);

    check_sizes();

    Info info;
    configure_info(info, model_id);

    LMFit lmfit
    (
        data_,
        weights_,
        info,
        initial_parameters_,
        parameters_to_fit_,
        user_info_,
        output_parameters_,
        output_states_,
        output_chi_squares_,
        output_n_iterations_
    ) ;
    lmfit.run(tolerance_);
}
/* -------------------------------------------------------------------------------------------------------
* from interface.cpp END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from gpufit.h START
------------------------------------------------------------------------------------------------------- */
/*
#ifdef __cplusplus
extern "C" {
#endif

VISIBLE int gpufit
(
    size_t n_fits,
    size_t n_points,
    REAL * data,
    REAL * weights,
    int model_id,
    REAL * initial_parameters,
    REAL tolerance,
    int max_n_iterations,
    int num_v_coefs, // NEW
    int * parameters_to_fit,
    int estimator_id,
    size_t user_info_size,
    char * user_info,
    REAL * output_parameters,
    int * output_states,
    REAL * output_chi_squares,
    int * output_n_iterations
) ;

VISIBLE int gpufit_cuda_interface
(
    size_t n_fits,
    size_t n_points,
    REAL * gpu_data,
    REAL * gpu_weights,
    int model_id,
    REAL tolerance,
    int max_n_iterations,
    int num_v_coefs, // NEW
    int * parameters_to_fit,
    int estimator_id,
    size_t user_info_size,
    char * gpu_user_info,
    REAL * gpu_fit_parameters,
    int * gpu_output_states,
    REAL * gpu_output_chi_squares,
    int * gpu_output_n_iterations
);

VISIBLE char const * gpufit_get_last_error() ;

//// returns 1 if cuda is available and 0 otherwise
//VISIBLE int gpufit_cuda_available();

VISIBLE int gpufit_get_cuda_version(int * runtime_version, int * driver_version);

VISIBLE int gpufit_portable_interface(int argc, void *argv[]);

#ifdef __cplusplus
}
#endif
*/
/* -------------------------------------------------------------------------------------------------------
* from gpufit.h END
------------------------------------------------------------------------------------------------------- */
/* -------------------------------------------------------------------------------------------------------
* from gpufit.cpp START
------------------------------------------------------------------------------------------------------- */
std::string last_error ;

int gpufit
(
    size_t n_fits,
    size_t n_points,
    REAL * data,
    REAL * weights,
    int model_id,
    REAL * initial_parameters,
    REAL tolerance,
    int max_n_iterations,
    int num_v_coefs, // NEW
    int * parameters_to_fit,
    int estimator_id,
    size_t user_info_size,
    char * user_info,
    REAL * output_parameters,
    int * output_states,
    REAL * output_chi_squares,
    int * output_n_iterations
)
try
{
    FitInterface fi(
        data,
        weights,
        n_fits,
        static_cast<int>(n_points),
        tolerance,
        max_n_iterations,
        num_v_coefs, // NEW
        static_cast<EstimatorID>(estimator_id),
        initial_parameters,
        parameters_to_fit,
        user_info,
        user_info_size,
        output_parameters,
        output_states,
        output_chi_squares,
        output_n_iterations,
        HOST);

    fi.fit(static_cast<ModelID>(model_id));

    return ReturnState::OK ;
}
catch( std::exception & exception )
{
    last_error = exception.what() ;

    return ReturnState::ERROR ;
}
catch( ... )
{
    last_error = "unknown error" ;

    return ReturnState::ERROR;
}

int gpufit_cuda_interface
(
    size_t n_fits,
    size_t n_points,
    REAL * gpu_data,
    REAL * gpu_weights,
    int model_id,
    REAL tolerance,
    int max_n_iterations,
    int num_v_coefs, // NEW
    int * parameters_to_fit,
    int estimator_id,
    size_t user_info_size,
    char * gpu_user_info,
    REAL * gpu_fit_parameters,
    int * gpu_output_states,
    REAL * gpu_output_chi_squares,
    int * gpu_output_n_iterations
)
try
{
    FitInterface fi(
        gpu_data,
        gpu_weights,
        n_fits,
        static_cast<int>(n_points),
        tolerance,
        max_n_iterations,
        num_v_coefs, // NEW
        static_cast<EstimatorID>(estimator_id),
        gpu_fit_parameters,
        parameters_to_fit,
        gpu_user_info,
        user_info_size,
        gpu_fit_parameters,
        gpu_output_states,
        gpu_output_chi_squares,
        gpu_output_n_iterations,
        DEVICE);

    fi.fit(static_cast<ModelID>(model_id));

    return ReturnState::OK;
}
catch (std::exception & exception)
{
    last_error = exception.what();

    return ReturnState::ERROR;
}
catch (...)
{
    last_error = "unknown error";

    return ReturnState::ERROR;
}

char const * gpufit_get_last_error()
{
    return last_error.c_str() ;
}

int gpufit_cuda_available()
{
	// Returns 1 if CUDA is available and 0 otherwise
	try
	{
		getDeviceCount();
		return 1;
	}
	catch (std::exception & exception)
	{
		last_error = exception.what();

		return 0;
	}
}

int gpufit_get_cuda_version(int * runtime_version, int * driver_version)
{
    try
    {
        hipRuntimeGetVersion(runtime_version);
        hipDriverGetVersion(driver_version);
        return ReturnState::OK;
    }
    catch (std::exception & exception)
    {
        last_error = exception.what();

        return ReturnState::ERROR;
    }
}

int gpufit_portable_interface(int argc, void *argv[])
{
// NOTE:
/*
0 int numberFits, 
1 int numberPoints, 
2 FloatBuffer data, 
3 FloatBuffer weights, 
4 int model_id, 
5 FloatBuffer initialParameters, 
6 float tolerance, 
7 int maxNumberIterations, 
8 int num_valid_coefs, 
9 IntBuffer parametersToFit, 
10 int estimatorID, 
11 int userInfoSize, 
12 FloatBuffer userInfo, 
13 FloatBuffer outputParameters, 
14 IntBuffer outputStates, 
15 FloatBuffer outputChiSquares, 
16 IntBuffer outputNumberIterations
*/
    return gpufit(
        *((size_t *) argv[0]),
        *((size_t *) argv[1]),
        (REAL *) argv[2],
        (REAL *) argv[3],
        *((int *) argv[4]),
        (REAL *) argv[5],
        *((REAL *) argv[6]),
        *((int *) argv[7]),
        *((int *) argv[8]),
        (int *) argv[9],
        *((int *) argv[10]),
        *((size_t *) argv[11]),
        (char *) argv[12],
        (REAL *) argv[13],
        (int *) argv[14],
        (REAL *) argv[15],
        (int *) argv[16]);

}
/* -------------------------------------------------------------------------------------------------------
* from gpufit.cpp END
------------------------------------------------------------------------------------------------------- */
/*
-------------------------------------------------------------------------------------------------------
* Calculate bleach correction input data
* from JCudaImageJExampleKernelcalcacf7.cu START
------------------------------------------------------------------------------------------------------- 
*/
__global__ void calc_data_bleach_correction(float* data, float* data1, int width, int height, int nopit, int ave)
{
    // function is an averaging step in temporal dimension for every ave number of points, prior to performing bleach correction fitting.

    int idx = blockIdx.x * blockDim.x + threadIdx.x, idy = blockIdx.y * blockDim.y + threadIdx.y;
    __syncthreads();
    
    if ( (idx < width) && (idy < height) )
    {
        for (int z1 = 0; z1 < nopit; z1++) {
            double sum1 = 0;

            for (int yy = z1 * ave; yy < (z1  + 1) * ave; yy++) {
                sum1 += (float) data[yy * width * height + idy * width + idx];
            } // for yy
            data1[idy*width*nopit + idx * nopit + z1] = sum1/ave;
        } // for z1
    } //if
}
/* -------------------------------------------------------------------------------------------------------
* from JCudaImageJExampleKernelcalcacf7.cu END
------------------------------------------------------------------------------------------------------- */
/*-------------------------------------------------------------------------------------------------------
* Calculate binning START
-------------------------------------------------------------------------------------------------------*/
__global__ void calc_binning(float* data, float* data1, int win_star, int hin_star, int w_temp, int h_temp, int framediff, int pixbinX, int pixbinY, int binningX, int binningY)
{
    // this function performs binning of spatial data.

    // NOTE: In the case overlap is OFF, we sill bin for every pixel, one pixel at a time.
    // This allows us to use cfXDistance and cfYDistance directly instead of translating these distances, which will be difficult.
 
    int idx = blockIdx.x * blockDim.x + threadIdx.x, idy = blockIdx.y * blockDim.y + threadIdx.y;
    __syncthreads();

    float sum = 0.0;    

    if ( (idx < w_temp) && (idy < h_temp) )
    {
        for (int t = 0; t < framediff; t++) {
            sum = 0.0;
            for (int i = 0; i < binningX; i++) {
                for (int j = 0; j < binningY; j++) {
                    sum += data[t * win_star * hin_star + (idy + j) * win_star + (idx + i)];  
                } // for j
            } // for i
            
            data1[t * w_temp * h_temp + idy * w_temp + idx] = sum;

        } // for t
    } // if
}

/*-------------------------------------------------------------------------------------------------------
* Calculate binning END
-------------------------------------------------------------------------------------------------------*/
/* -------------------------------------------------------------------------------------------------------
* from com_github_gpufit_Gpufit.cpp START
* NOTE: creates the gpufitJNI.dll. File is located at /Gpufit/Gpufit/java/adapter/
------------------------------------------------------------------------------------------------------- */
void * buffer_address(JNIEnv * env, jobject buffer)
{
    if (buffer == 0)
    {
        return 0;
    }
    else
    {
        return env->GetDirectBufferAddress(buffer);
    }
}

/*
* Calls gpufit(), no consistency checks on this side.
*
* Class:     gpufitImFCS_GpufitImFCS
* Method:    fit
* Signature: (IILjava/nio/FloatBuffer;Ljava/nio/FloatBuffer;ILjava/nio/FloatBuffer;FILjava/nio/IntBuffer;IILjava/nio/ByteBuffer;Ljava/nio/FloatBuffer;Ljava/nio/IntBuffer;Ljava/nio/FloatBuffer;Ljava/nio/IntBuffer;)I
*/
jint JNICALL Java_gpufitImFCS_GpufitImFCS_fit(JNIEnv * env, jclass cls, jint number_fits, jint number_points, jobject data_buffer, jobject weights_buffer, jint model_id, jobject initial_parameter_buffer, jfloat tolerance, jint max_number_iterations, jint num_valid_coefs, jobject paramters_to_fit_buffer, jint estimator_id, jint user_info_size, jobject user_info_buffer, jobject output_parameters_buffer, jobject output_states_buffer, jobject output_chi_squares_buffer, jobject output_number_iterations_buffer)
{
    // get pointer to buffers
    REAL * data = (REAL *)buffer_address(env, data_buffer);
    REAL * weights = (REAL *)buffer_address(env, weights_buffer);
    REAL * initial_parameters = (REAL *)buffer_address(env, initial_parameter_buffer);
    int * parameters_to_fit = (int *)buffer_address(env, paramters_to_fit_buffer);
    char * user_info = (char *)buffer_address(env, user_info_buffer);
    REAL * output_parameters = (REAL *)buffer_address(env, output_parameters_buffer);
    int * output_states = (int *)buffer_address(env, output_states_buffer);
    REAL * output_chi_squares = (REAL *)buffer_address(env, output_chi_squares_buffer);
    int * output_number_iterations = (int *)buffer_address(env, output_number_iterations_buffer);

    // call to gpufit
    // NOTE: Added num_valid_coefs
    int status = gpufit(number_fits, number_points, data, weights, model_id, initial_parameters, tolerance, max_number_iterations, num_valid_coefs, parameters_to_fit, estimator_id, user_info_size, user_info, output_parameters, output_states, output_chi_squares, output_number_iterations);

// lmfit_cuda

    return status;
}

/*
* Calls gpufit_get_last_error()
*
* Class:     gpufitImFCS_GpufitImFCS
* Method:    getLastError
* Signature: ()Ljava/lang/String;
*/
jstring JNICALL Java_gpufitImFCS_GpufitImFCS_getLastError(JNIEnv * env, jclass cls)
{
    char const * error = gpufit_get_last_error();
    return env->NewStringUTF(error);
}

/*
* Calls gpufit_cuda_available()
*
* Class:     gpufitImFCS_GpufitImFCS
* Method:    isCudaAvailableInt
* Signature: ()Z
*/
jboolean JNICALL Java_gpufitImFCS_GpufitImFCS_isCudaAvailableInt(JNIEnv * env, jclass cls)
{
    return gpufit_cuda_available() == 1 ? JNI_TRUE : JNI_FALSE;
}

/*
* Calls gpufit_get_cuda_version()
*
* Class:     gpufitImFCS_GpufitImFCS
* Method:    getCudaVersionAsArray
* Signature: ()[I
*/
jintArray JNICALL Java_gpufitImFCS_GpufitImFCS_getCudaVersionAsArray(JNIEnv * env, jclass cls)
{
    int runtime_version, driver_version;
    if (gpufit_get_cuda_version(&runtime_version, &driver_version) == ReturnState::OK)
    {
        // create int[2] in Java and fill with values
        jintArray array = env->NewIntArray(2);
        jint fill[2];
        fill[0] = runtime_version;
        fill[1] = driver_version;
        env->SetIntArrayRegion(array, 0, 2, fill);
        return array;
    }
    else
    {
        return 0;
    }
}

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    resetGPU
 * Signature: ()V
 */
void JNICALL Java_gpufitImFCS_GpufitImFCS_resetGPU(JNIEnv * env, jclass cls)
{
    try{
        hipDeviceReset();
    } catch(std::runtime_error & e) {
        // see: https://www.rgagnon.com/javadetails/java-0323.html
        jclass Exception = env->FindClass("java/lang/Exception");
        env->ThrowNew(Exception, e.what());
    }
}

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    calcDataBleachCorrection
 * Signature: ([F[FLgpufitImFCS/GpufitImFCS/ACFParameters;)V
 */
void JNICALL Java_gpufitImFCS_GpufitImFCS_calcDataBleachCorrection(JNIEnv * env, jclass cls, jfloatArray pixels, jfloatArray outdata, jobject ACFInputParams)
{
    size_t SIZEFLOAT = sizeof(float);

    // input arrays required for calculations.
    jfloat *Cpixels;
    float *d_Cpixels;

    //output data
    float *Coutput;
    float *d_Coutput;

    try{
        Cpixels = env->GetFloatArrayElements(pixels, NULL);        

        // get parameters from the ACFInputParams object
        // we need width, height, cfXDistancegpu, cfYDistancegpu, nopit, ave
        // we also need firstframe and lastframe for setting blockSize and gridSize
        jclass ACFInputParamsCls = env->GetObjectClass(ACFInputParams);

        jfieldID w_tempId = env->GetFieldID(ACFInputParamsCls, "w_temp", "I");
        jfieldID h_tempId = env->GetFieldID(ACFInputParamsCls, "h_temp", "I");
        jfieldID firstframeId = env->GetFieldID(ACFInputParamsCls, "firstframe", "I");
        jfieldID lastframeId = env->GetFieldID(ACFInputParamsCls, "lastframe", "I");
        jfieldID nopitId = env->GetFieldID(ACFInputParamsCls, "nopit", "I");
        jfieldID aveId = env->GetFieldID(ACFInputParamsCls, "ave", "I");

        jint w_temp = env->GetIntField(ACFInputParams, w_tempId);
        jint h_temp = env->GetIntField(ACFInputParams, h_tempId);
        jint firstframe = env->GetIntField(ACFInputParams, firstframeId);
        jint lastframe = env->GetIntField(ACFInputParams, lastframeId);
        jint nopit = env->GetIntField(ACFInputParams, nopitId);
        jint ave = env->GetIntField(ACFInputParams, aveId);

        // blockSize and gridSize
        int framediff = lastframe - firstframe + 1;
        int BLKSIZEXY = 16;
        int a = ( w_temp > h_temp) ? w_temp : h_temp;
        int GRIDSIZEXY = (a + BLKSIZEXY -1) / BLKSIZEXY;

        dim3 blockSize(BLKSIZEXY, BLKSIZEXY, 1);
        dim3 gridSize(GRIDSIZEXY, GRIDSIZEXY, 1);

        // Allocate memory on GPU
        size_t size = w_temp * h_temp * framediff * SIZEFLOAT;
        hipMalloc((void **)&d_Cpixels, size);

        // Allocate memory for Coutput and d_Coutput
        unsigned int sizeA = w_temp * h_temp * nopit;
        size_t size1 = sizeA * SIZEFLOAT;

        Coutput = (float *)malloc(size1);
        hipMalloc((void **)&d_Coutput, size1);

        // Copy to GPU
        CUDA_CHECK_STATUS(hipMemcpy(d_Cpixels, Cpixels, size, hipMemcpyHostToDevice));

        hipStream_t stream;
        CUDA_CHECK_STATUS(hipStreamCreate( &stream ));

        calc_data_bleach_correction<<<gridSize, blockSize, 0, stream>>>(d_Cpixels, d_Coutput, w_temp, h_temp, nopit, ave);

        hipDeviceSynchronize();
        CUDA_CHECK_STATUS(hipGetLastError());

        // copy memory from device to host
        CUDA_CHECK_STATUS(hipMemcpy(Coutput, d_Coutput, size1, hipMemcpyDeviceToHost));

        CUDA_CHECK_STATUS(hipStreamDestroy( stream ));

        //CUDA release memory
        hipFree(d_Cpixels); hipFree(d_Coutput);

        hipDeviceReset();

        // copy values to Java output arrays.
        env->SetFloatArrayRegion(outdata, 0 , sizeA, Coutput);  

        //free pointers
        free(Coutput);

        // release resources
        env->ReleaseFloatArrayElements(pixels, Cpixels, 0);        
       
    } catch(std::runtime_error & e) {

        //CUDA release memory
        hipFree(d_Cpixels); hipFree(d_Coutput);

        //free pointers
        free(Coutput);

        // release resources
        env->ReleaseFloatArrayElements(pixels, Cpixels, 0);

        // see: https://www.rgagnon.com/javadetails/java-0323.html
        jclass Exception = env->FindClass("java/lang/Exception");
        env->ThrowNew(Exception, e.what());
    }

    return;
}

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    isBinningMemorySufficient
 * Signature: (LgpufitImFCS/GpufitImFCS/ACFParameters;)Z
 */
jboolean JNICALL Java_gpufitImFCS_GpufitImFCS_isBinningMemorySufficient(JNIEnv * env, jclass cls, jobject ACFInputParams)
{

  try {
      unsigned int SIZEFLOAT = sizeof(float);

      // get parameters from the ACFInputParams object
      jclass ACFInputParamsCls = env->GetObjectClass(ACFInputParams);
      jfieldID win_starId = env->GetFieldID(ACFInputParamsCls, "win_star", "I");
      jfieldID hin_starId = env->GetFieldID(ACFInputParamsCls, "hin_star", "I");
      jfieldID w_tempId = env->GetFieldID(ACFInputParamsCls, "w_temp", "I");
      jfieldID h_tempId = env->GetFieldID(ACFInputParamsCls, "h_temp", "I");
      jfieldID firstframeId = env->GetFieldID(ACFInputParamsCls, "firstframe", "I");
      jfieldID lastframeId = env->GetFieldID(ACFInputParamsCls, "lastframe", "I");

      jint win_star = env->GetIntField(ACFInputParams, win_starId);
      jint hin_star = env->GetIntField(ACFInputParams, hin_starId);
      jint w_temp = env->GetIntField(ACFInputParams, w_tempId);
      jint h_temp = env->GetIntField(ACFInputParams, h_tempId);
      jint firstframe = env->GetIntField(ACFInputParams, firstframeId);
      jint lastframe = env->GetIntField(ACFInputParams, lastframeId);

      // sanity check if memory on GPU is sufficient for binning
      std::size_t this_free_bytes;
      std::size_t this_total_bytes;    
      int framediff = lastframe - firstframe + 1;
      double maxmemory = (double) (win_star * hin_star + w_temp * h_temp) * framediff * SIZEFLOAT;
      CUDA_CHECK_STATUS(hipMemGetInfo(&this_free_bytes, &this_total_bytes));

      return (maxmemory > double(this_free_bytes) * 0.9) ? JNI_FALSE : JNI_TRUE;

  } catch (std::runtime_error & e)  {
      jclass Exception = env->FindClass("java/lang/Exception");
      env->ThrowNew(Exception, e.what());
      return JNI_FALSE;
  }

}

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    calcBinning
 * Signature: ([F[FLgpufitImFCS/GpufitImFCS/ACFParameters;)V
 */
void JNICALL Java_gpufitImFCS_GpufitImFCS_calcBinning(JNIEnv * env, jclass cls, jfloatArray indata, jfloatArray outdata, jobject ACFInputParams)
{
    size_t SIZEFLOAT = sizeof(float);

    // input arrays required for calculations.
    jfloat *Cindata;
    float *d_Cindata;

    //output data
    float *Coutput;
    float *d_Coutput;

    try{
        Cindata = env->GetFloatArrayElements(indata, NULL);        

        // get parameters from the ACFInputParams object
        // we need w_temp, h_temp, binningX, binningY
        // we also need firstframe and lastframe for setting blockSize and gridSize
        jclass ACFInputParamsCls = env->GetObjectClass(ACFInputParams);
        jfieldID win_starId = env->GetFieldID(ACFInputParamsCls, "win_star", "I");
        jfieldID hin_starId = env->GetFieldID(ACFInputParamsCls, "hin_star", "I");
        jfieldID w_tempId = env->GetFieldID(ACFInputParamsCls, "w_temp", "I");
        jfieldID h_tempId = env->GetFieldID(ACFInputParamsCls, "h_temp", "I");
        jfieldID pixbinXId = env->GetFieldID(ACFInputParamsCls, "pixbinX", "I");
        jfieldID pixbinYId = env->GetFieldID(ACFInputParamsCls, "pixbinY", "I");
        jfieldID binningXId = env->GetFieldID(ACFInputParamsCls, "binningX", "I");
        jfieldID binningYId = env->GetFieldID(ACFInputParamsCls, "binningY", "I");
        jfieldID firstframeId = env->GetFieldID(ACFInputParamsCls, "firstframe", "I");
        jfieldID lastframeId = env->GetFieldID(ACFInputParamsCls, "lastframe", "I");

        jint win_star = env->GetIntField(ACFInputParams, win_starId);
        jint hin_star = env->GetIntField(ACFInputParams, hin_starId);
        jint w_temp = env->GetIntField(ACFInputParams, w_tempId);
        jint h_temp = env->GetIntField(ACFInputParams, h_tempId);
        jint pixbinX = env->GetIntField(ACFInputParams, pixbinXId);
        jint pixbinY = env->GetIntField(ACFInputParams, pixbinYId);
        jint binningX = env->GetIntField(ACFInputParams, binningXId);
        jint binningY = env->GetIntField(ACFInputParams, binningYId);
        jint firstframe = env->GetIntField(ACFInputParams, firstframeId);
        jint lastframe = env->GetIntField(ACFInputParams, lastframeId);

        // blockSize and gridSize
        int framediff = lastframe - firstframe + 1;
        int BLKSIZEXY = 16;
        int a = ( w_temp > h_temp) ? w_temp : h_temp;
        int GRIDSIZEXY = (a + BLKSIZEXY - 1) / BLKSIZEXY;

        dim3 blockSizeBin(BLKSIZEXY, BLKSIZEXY, 1);
        dim3 gridSizeBin(GRIDSIZEXY, GRIDSIZEXY, 1);

        // Allocate memory on GPU
        size_t size = win_star * hin_star * framediff * SIZEFLOAT;
        hipMalloc((void **)&d_Cindata, size);

        // Allocate memory for Coutput and d_Coutput
        unsigned int sizeA = w_temp * h_temp * framediff;
        size_t size1 = sizeA * SIZEFLOAT;
        Coutput = (float *)malloc(size1);
        hipMalloc((void **)&d_Coutput, size1);

        // Copy to GPU
        CUDA_CHECK_STATUS(hipMemcpy(d_Cindata, Cindata, size, hipMemcpyHostToDevice));

        hipStream_t stream;
        CUDA_CHECK_STATUS(hipStreamCreate( &stream ));

        calc_binning<<<gridSizeBin, blockSizeBin, 0, stream>>>(d_Cindata, d_Coutput, win_star, hin_star, w_temp, h_temp, framediff, pixbinX, pixbinY, binningX, binningY);

        hipDeviceSynchronize();
        CUDA_CHECK_STATUS(hipGetLastError());

        // copy memory from device to host
        CUDA_CHECK_STATUS(hipMemcpy(Coutput, d_Coutput, size1, hipMemcpyDeviceToHost));

        CUDA_CHECK_STATUS(hipStreamDestroy( stream ));

        //CUDA release memory
        hipFree(d_Cindata); hipFree(d_Coutput);

        hipDeviceReset();

        // copy values to Java output arrays.
        env->SetFloatArrayRegion(outdata, 0 , sizeA, Coutput);  
 
        //free pointers
        free(Coutput);

        // release resources
        env->ReleaseFloatArrayElements(indata, Cindata, 0);

    } catch(std::runtime_error & e) {
        //CUDA release memory
        hipFree(d_Cindata); hipFree(d_Coutput);

        //free pointers
        free(Coutput);

        // release resources
        env->ReleaseFloatArrayElements(indata, Cindata, 0);

        // see: https://www.rgagnon.com/javadetails/java-0323.html
        jclass Exception = env->FindClass("java/lang/Exception");
        env->ThrowNew(Exception, e.what());
    }

    return;

}

/*
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    isACFmemorySufficient
 * Signature: (LgpufitImFCS/GpufitImFCS/ACFParameters;)Z
 */
jboolean JNICALL Java_gpufitImFCS_GpufitImFCS_isACFmemorySufficient(JNIEnv * env, jclass cls, jobject ACFInputParams)
{
  try{
      unsigned int SIZEINT = sizeof(int);
      unsigned int SIZEFLOAT = sizeof(float);
      unsigned int SIZEDOUBLE = sizeof(double);

      double totalmemoryAll = 0.0;
      double totalmemoryCalc3 = 0.0;
      double totalmemoryCalc2 = 0.0;

      // get parameters that are required for the ACF calculations from the ACFInputParams object
      jclass ACFInputParamsCls = env->GetObjectClass(ACFInputParams);

      jfieldID widthId = env->GetFieldID(ACFInputParamsCls, "width", "I");
      jfieldID heightId = env->GetFieldID(ACFInputParamsCls, "height", "I");
      jfieldID w_tempId = env->GetFieldID(ACFInputParamsCls, "w_temp", "I");
      jfieldID h_tempId = env->GetFieldID(ACFInputParamsCls, "h_temp", "I");
      jfieldID firstframeId = env->GetFieldID(ACFInputParamsCls, "firstframe", "I");
      jfieldID lastframeId = env->GetFieldID(ACFInputParamsCls, "lastframe", "I");
      jfieldID cfXDistanceId = env->GetFieldID(ACFInputParamsCls, "cfXDistance", "I");
      jfieldID cfYDistanceId = env->GetFieldID(ACFInputParamsCls, "cfYDistance", "I");
//      jfieldID correlatorpId = env->GetFieldID(ACFInputParamsCls, "correlatorp", "D");
//      jfieldID correlatorqId = env->GetFieldID(ACFInputParamsCls, "correlatorq", "D");
      jfieldID frametimeId = env->GetFieldID(ACFInputParamsCls, "frametime", "D");
      jfieldID backgroundId = env->GetFieldID(ACFInputParamsCls, "background", "I");
      jfieldID mtab1Id = env->GetFieldID(ACFInputParamsCls, "mtab1", "D");
      jfieldID mtabchanumminus1Id = env->GetFieldID(ACFInputParamsCls, "mtabchanumminus1", "D");
      jfieldID sampchanumminus1Id = env->GetFieldID(ACFInputParamsCls, "sampchanumminus1", "D");
      jfieldID chanumId = env->GetFieldID(ACFInputParamsCls, "chanum", "I");
      jfieldID isNBcalculationId = env->GetFieldID(ACFInputParamsCls, "isNBcalculation", "Z");
      jfieldID bleachcorr_gpuId = env->GetFieldID(ACFInputParamsCls, "bleachcorr_gpu", "Z");
      jfieldID bleachcorr_orderId = env->GetFieldID(ACFInputParamsCls, "bleachcorr_order", "I");

      jint width = env->GetIntField(ACFInputParams, widthId);
      jint height = env->GetIntField(ACFInputParams, heightId);
      jint w_temp = env->GetIntField(ACFInputParams, w_tempId);
      jint h_temp = env->GetIntField(ACFInputParams, h_tempId);
      jint firstframe = env->GetIntField(ACFInputParams, firstframeId);
      jint lastframe = env->GetIntField(ACFInputParams, lastframeId);
      jint cfXDistance = env->GetIntField(ACFInputParams, cfXDistanceId);
      jint cfYDistance = env->GetIntField(ACFInputParams, cfYDistanceId);
//      jdouble correlatorpdbl = env->GetDoubleField(ACFInputParams, correlatorpId);
//      jdouble correlatorqdbl = env->GetDoubleField(ACFInputParams, correlatorqId);
      jdouble frametime = env->GetDoubleField(ACFInputParams, frametimeId);
      jint background = env->GetIntField(ACFInputParams, backgroundId);
      jdouble mtab1 = env->GetDoubleField(ACFInputParams, mtab1Id); // mtab[1], used to calculate blocknumgpu.
      jdouble mtabchanumminus1 = env->GetDoubleField(ACFInputParams, mtabchanumminus1Id); // mtab[chanum-1], used to calculate pnumgpu[counter_indexarray]
      jdouble sampchanumminus1 = env->GetDoubleField(ACFInputParams, sampchanumminus1Id); // samp[chanum-1], used to calculate pnumgpu[counter_indexarray]
      jint chanum = env->GetIntField(ACFInputParams, chanumId);
      jboolean isNBcalculation = env->GetBooleanField(ACFInputParams, isNBcalculationId);      
      jboolean bleachcorr_gpu = env->GetBooleanField(ACFInputParams, bleachcorr_gpuId);      
      jint bleachcorr_order = env->GetIntField(ACFInputParams, bleachcorr_orderId);

      // initialize parameters
//      int correlatorp = (int) correlatorpdbl;
//      int correlatorq = (int) correlatorqdbl;
      int framediff = lastframe - firstframe + 1;
      unsigned long size = w_temp * h_temp * framediff * SIZEFLOAT;
      unsigned long size1 = width * height * chanum * SIZEDOUBLE;
      unsigned long size2 = framediff * width * height * SIZEFLOAT;
      unsigned long sizeblockvararray = chanum * width * height * SIZEDOUBLE;
          
      int blocknumgpu = (int) (floor(log(mtab1)/log(2)) - 2);

      // dynamic memory allocation and/or initialization
      //------------------ common parameters ---------------------------
      totalmemoryAll = totalmemoryAll + (double) size1; //Cpixels1
      totalmemoryAll = totalmemoryAll + (double) size2; //prod
      totalmemoryAll = totalmemoryAll + (double) size; //pixels
      totalmemoryAll = totalmemoryAll + (double) (chanum * SIZEDOUBLE); //samp
      totalmemoryAll = totalmemoryAll + (double) (chanum * SIZEINT); //lag
      totalmemoryAll = totalmemoryAll + (double) width * height * bleachcorr_order * SIZEDOUBLE; 
      // totalmemoryAll = totalmemoryAll + (double) size1; // Cblocked1D copies Cpixels1 array after calcacf3 calculation and not required GPU memory.

      //------------------ calcacf3 ---------------------------
      if (!isNBcalculation) {
          totalmemoryCalc3 = totalmemoryCalc3 + (double) (blocknumgpu * SIZEDOUBLE); //prodnum
          totalmemoryCalc3 = totalmemoryCalc3 + (double) sizeblockvararray; //blocksd
          totalmemoryCalc3 = totalmemoryCalc3 + (double) (blocknumgpu*width*height*SIZEDOUBLE); //upper
          totalmemoryCalc3 = totalmemoryCalc3 + (double) (blocknumgpu*width*height*SIZEDOUBLE); //lower
          totalmemoryCalc3 = totalmemoryCalc3 + (double) ((blocknumgpu-1)*width*height*SIZEINT); //crt
          totalmemoryCalc3 = totalmemoryCalc3 + (double) ((blocknumgpu-2)*width*height*SIZEINT); //cr12
          totalmemoryCalc3 = totalmemoryCalc3 + (double) ((blocknumgpu-2)*width*height*SIZEINT); //cr3
          totalmemoryCalc3 = totalmemoryCalc3 + (double) ((blocknumgpu-1)*width*height*SIZEINT); //diffpos
          totalmemoryCalc3 = totalmemoryCalc3 + (double) (blocknumgpu*width*height*SIZEDOUBLE); //varblock0
          totalmemoryCalc3 = totalmemoryCalc3 + (double) (blocknumgpu*width*height*SIZEDOUBLE); //varblock1
          totalmemoryCalc3 = totalmemoryCalc3 + (double) (blocknumgpu*width*height*SIZEDOUBLE); //varblock2
      }

      //------------------ calcacf2 ---------------------------
      totalmemoryCalc2 = totalmemoryCalc2 + (double) (chanum*width*height*SIZEINT); //prodnumarray
      totalmemoryCalc2 = totalmemoryCalc2 + (double) (width*height*SIZEINT); //indexarray
      totalmemoryCalc2 = totalmemoryCalc2 + (double) sizeblockvararray; //Cblockvararray
      totalmemoryCalc2 = totalmemoryCalc2 + (double) sizeblockvararray; //blocksdarray
      totalmemoryCalc2 = totalmemoryCalc2 + (double) (width*height*SIZEINT); //pnumgpu
 
      //------------------ calculation of N & B in calcacf2 --------------------------
      if (isNBcalculation) {
          totalmemoryCalc2 = totalmemoryCalc2 + (double) ( width * height * SIZEDOUBLE * 3); //NBmeanGPU, NBmean2GPU, NBcovarianceGPU
      }

      // sanity check if memory on GPU is sufficient for calcacf3 and calcacf2
      // NOTE calcacf3 will run first. Memory of parameters related to calcacf3 only will be released after completion of calcacf3.
      std::size_t this_free_bytes;
      std::size_t this_total_bytes;
      double maxmemory = (totalmemoryCalc3 > totalmemoryCalc2)? totalmemoryCalc3 : totalmemoryCalc2;
      maxmemory = maxmemory + totalmemoryAll;
      CUDA_CHECK_STATUS(hipMemGetInfo(&this_free_bytes, &this_total_bytes));

      return (maxmemory > double(this_free_bytes) * 0.9) ? JNI_FALSE : JNI_TRUE;

  } catch (std::runtime_error & e)  {
    jclass Exception = env->FindClass("java/lang/Exception");
    env->ThrowNew(Exception, e.what());
    return JNI_FALSE;
  }

}

/* ------------------------------------------
AUTOCORRELATION SINGLE DIMENSION ARRAY CALCULATION START
NOTE: SEE JCudaImageJExampleKernelcalcacf2.cu
------------------------------------------ */
__global__ void bleachcorrection(float * data, int w_temp, int h_temp, int d, int bleachcorr_order, double frametimegpu, double* bleachcorr_params)
{
    // function performs polynomial bleach correction given polynomial order and coefficients. It is done prior to calcacf3, calcacf2a and calcacf2b.

    int idx = blockIdx.x * blockDim.x + threadIdx.x, idy = blockIdx.y * blockDim.y + threadIdx.y;
    __syncthreads();
    if ( (idx < w_temp) && (idy < h_temp) ) {
        for (int i = 0; i < d; i++) {
            float corfunc = 0;
            for (int ii = 0; ii < bleachcorr_order; ii++) {
                  corfunc += bleachcorr_params[(idy*w_temp +idx)*bleachcorr_order + ii] * powf((float)frametimegpu * (i + 0.5), (float)ii);
            } // for ii
            
            float res0 = bleachcorr_params[(idy*w_temp + idx) * bleachcorr_order];

	    data[i*w_temp*h_temp + idy*w_temp + idx] = data[i*w_temp*h_temp + idy*w_temp + idx] / sqrtf(corfunc / res0) + res0 * (1 - sqrtf(corfunc / res0));
            __syncthreads();
        } // for i
    } // if ((idx < w_temp) && (idy < h_temp))
} //bleachcorrection function

__device__ unsigned int countx = 0;
__device__ unsigned int county = 0;
__shared__ bool isLastBlockDone;
__global__ void calcacf2a(float* data, int w_temp, int h_temp, int numbin)
{
    // function calculates the arrays according to different time bins in different parts of the correlation function

    int idx = blockIdx.x * blockDim.x + threadIdx.x, idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    __syncthreads();
    if ( (idx < w_temp) && (idy < h_temp) ) {
        // And correct the number Of actual data points accordingly
        for (int y = 0; y < numbin; y++){ // if yes, bin the data according to the width of the current channel
            data[y*w_temp*h_temp + idy*w_temp + idx] = data[2 * y*w_temp*h_temp + idy*w_temp + idx] + data[(2 * y + 1)*w_temp*h_temp + idy*w_temp + idx];
            __syncthreads();            
        } // for int y = 0 
    }
}

__global__ void calcacf2b(float* data, int cfXDistancegpu, int cfYDistancegpu, int w, int h, int w_temp, int h_temp, int pixbinX, int pixbinY, double* data1, float* prod, int*laggpu, int* prodnumarray, int* indexarray, double* blockvararray, double* sdarray, int* pnumgpu, int x, int numbin, int currentIncrement, int ctbin, bool isNBcalculation, double* NBmeanGPU, double* NBcovarianceGPU)
{
    // function calculates the value of the auto or cross-correlation at every lag time. This function also performs the G1 analysis in N and B calculation.

    int del;				// delay Or correlation time expressed In lags
    double sumprod = 0.0;		// sum of all intensity products; divide by num to get the average <i(n)i(n+del)>

    int idx = blockIdx.x * blockDim.x + threadIdx.x, idy = blockIdx.y * blockDim.y + threadIdx.y;
    double temp1=0.0, temp2=0.0;

    __syncthreads();
    if ( (idx < w) && (idy < h) ){

        del = laggpu[x] / currentIncrement;
	prodnumarray[x*w*h + idy*w + idx] = numbin - del;

	temp1 = 0.0;
	temp2 = 0.0;
	    
        for (int y = 0; y < prodnumarray[x*w*h + idy*w + idx]; y++){ // calculate the ...
	    temp1 += data[y*w_temp*h_temp + idy*pixbinY*w_temp +  idx*pixbinX];
            temp2 += data[(y + del)*w_temp*h_temp + (idy*pixbinY + cfYDistancegpu)*w_temp + (idx*pixbinX + cfXDistancegpu)];
        }

	temp1 /= prodnumarray[x*w*h + idy*w + idx]; // calculate average of direct and delayed monitor, i.e. the average intensity <n(0)> and <n(tau)>
        temp2 /= prodnumarray[x*w*h + idy*w + idx];
	sumprod = 0.0;

	for (int y = 0; y < prodnumarray[x*w*h + idy*w + idx]; y++){ // calculate the correlation
            if (isNBcalculation) {
               prod[y*w*h + idy*w + idx] =  data[y*w_temp*h_temp + idy*pixbinY*w_temp + idx*pixbinX] * data[(y + del)*w_temp*h_temp + (idy*pixbinY + cfYDistancegpu)*w_temp + (idx*pixbinX + cfXDistancegpu)];
           
            } else {
                prod[y*w*h + idy*w + idx] =  data[y*w_temp*h_temp + idy*pixbinY*w_temp + idx*pixbinX] * data[(y + del)*w_temp*h_temp + (idy*pixbinY + cfYDistancegpu)*w_temp + (idx*pixbinX + cfXDistancegpu)] - temp2 * data[y*w_temp*h_temp + idy*pixbinY*w_temp +  idx*pixbinX] - temp1 * data[(y + del)*w_temp*h_temp + (idy*pixbinY + cfYDistancegpu)*w_temp + (idx*pixbinX + cfXDistancegpu)] + temp1*temp2;
            }
            sumprod += prod[y*w*h + idy*w + idx];	
        }

        if (isNBcalculation) {
            NBmeanGPU[idy * w + idx] = temp1;
            NBcovarianceGPU[idy * w + idx] = sumprod/prodnumarray[x*w*h + idy*w + idx] -temp1*temp2;
        }

	__syncthreads();
 
        if (!isNBcalculation) {
            data1[x*w*h + idy*w + idx] = sumprod / (prodnumarray[x*w*h + idy*w + idx] * temp1 * temp2);
	    __syncthreads();

            sumprod = 0.0;
            double sumprod2 = 0.0;	// sum of all intensity products squared; divide by num to get the average <(i(n)i(n+del))^2>
            int binct = indexarray[idy*w + idx]-ctbin;
            double tempvariable =0.0;
            for (int y = 1; y <=binct; y++) {
                prodnumarray[x*w*h + idy*w + idx] = (int)floor((double)prodnumarray[x*w*h + idy*w + idx] / 2.0);
							 
                for (int z = 0; z < prodnumarray[x*w*h + idy*w + idx]; z++) {
	            prod[z*w*h + idy*w + idx]  = (prod[2 * z*w*h + idy*w + idx] + prod[(2 * z + 1)*w*h + idy*w + idx]) / 2.0;
                    __syncthreads();
	        }
            }
 			 	
	    for (int z = 0; z < pnumgpu[idy*w + idx]; z++) {
                tempvariable =  prod[z*w*h + idy*w + idx];
                sumprod += tempvariable; // calculate the sum of prod, i.e. the raw correlation value ...
                sumprod2 += powf(tempvariable, 2.0); // ... and the sum of the squares
            }

            blockvararray[x*w*h + idy*w + idx] = (sumprod2 / pnumgpu[idy*w + idx] - powf(sumprod / pnumgpu[idy*w + idx], 2.0)) / ((pnumgpu[idy*w + idx] - 1) * powf(temp1 * temp2, 2.0));
					
            sdarray[x*w*h + idy*w + idx] = sqrt(blockvararray[x*w*h + idy*w + idx]);
       } // if (!isNBcalculation)

    } // if ((idx < w) && (idy < h))
} //calcacf2b

/* ------------------------------------------
NOTE: SEE JCudaImageJExampleKernelcalcacf3.cu
------------------------------------------ */
__global__ void calcacf3(float* data, int cfXDistancegpu, int cfYDistancegpu, int blocklag, int w, int h, int w_temp, int h_temp, int pixbinX, int pixbinY, int d, int correlatorp, int correlatorq, int chanum, double frametimegpu, double* data1, float* prod, double* prodnum, double* blocksd, double* upper, double* lower, int* crt, int* cr12, int* cr3, int* diffpos, double* varblock0, double* varblock1, double* varblock2, double* sampgpu, int*laggpu)
{
    // this function calculates the block transformation values of the intensity.

    int blocknumgpu = (int)floor(log((double)d - 1.0) / log(2.0)) - 2;
    int numbin = d;		// number Of data points When they are binned
    int del;				// delay Or correlation time expressed In lags
    int currentIncrement = blocklag;
    double sumprod = 0.0;		// sum of all intensity products; divide by num to get the average <i(n)i(n+del)>
    double sumprod2 = 0.0;	// sum of all intensity products squared; divide by num to get the average <(i(n)i(n+del))^2>
    double directm = 0.0;		// direct monitor required for ACF normalization
    double delayedm = 0.0;
    int ind = 0;
    int last0 = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x, idy = blockIdx.y * blockDim.y + threadIdx.y;
    int blockIndS = 0;

    __syncthreads();
    if ((idx < w) && (idy < h)){

        int  x = 1;
	del = laggpu[x] / currentIncrement; // calculate the delay, i.e. the correlation time
	for (int y = 0; y < numbin - del; y++) { // calculate the ...
            directm += data[y*w_temp*h_temp + idy*pixbinY*w_temp + idx*pixbinX]; // direct And ...
	    delayedm += data[(y + del)*w_temp*h_temp + (idy*pixbinY + cfYDistancegpu)*w_temp + (idx*pixbinX + cfXDistancegpu)]; // delayed monitor
	}
	prodnum[0] = numbin - del; // number Of correlation products
	directm /= prodnum[0]; // calculate average Of direct And delayed monitor, 
	delayedm /= prodnum[0]; // i.e. the average intesity <n(0)> And <n(tau)>

	for (int y = 0; y < prodnum[0]; y++) { // calculate the correlation
	    prod[y*w*h + idy*w + idx] = data[y*w_temp*h_temp + idy*pixbinY*(w + cfXDistancegpu) + idx*pixbinX] * data[(y + del)*w_temp*h_temp + (idy*pixbinY + cfYDistancegpu)*w_temp + (idx*pixbinX + cfXDistancegpu)] - delayedm * data[y*w_temp*h_temp + idy*pixbinY*w_temp + idx*pixbinX] - directm * data[(y + del)*w_temp*h_temp + (idy*pixbinY + cfYDistancegpu)*w_temp + (idx*pixbinX + cfXDistancegpu)] + delayedm * directm;
            __syncthreads();
	    sumprod += prod[y*w*h + idy*w + idx]; // calculate the sum Of prod, i.e. the raw correlation value ...
	    sumprod2 += powf(prod[y*w*h + idy*w + idx], 2.0); // ... And the sum Of the squares
	}

	varblock0[idy*w + idx] = currentIncrement * frametimegpu; // the time Of the block curve
	varblock1[idy*w + idx] = (sumprod2 / prodnum[0] - powf(sumprod / prodnum[0], 2.0)) / (prodnum[0] * powf(directm * delayedm, 2.0));

	for (int y = 1; y < blocknumgpu; y++) { // perform blocking operations
	    prodnum[y] = (int)floor((double)prodnum[y - 1] / 2);	// the number Of samples For the blocking curve decreases by a factor 2 With every Step
	    sumprod = 0;
	    sumprod2 = 0;
	    for (int z = 0; z < prodnum[y]; z++) { // bin the correlation data And calculate the blocked values for the SD
	        prod[z*w*h + idy*w + idx] = (prod[2 * z*w*h + idy*w + idx] + prod[(2 * z + 1)*w*h + idy*w + idx]) / 2;
                __syncthreads();
		sumprod += prod[z*w*h + idy*w + idx];
		sumprod2 += powf(prod[z*w*h + idy*w + idx], 2.0);
	    }
	
            // This is the correct one
	    varblock0[y*w*h + idy*w + idx] = (currentIncrement * powf(2, (double)y)) * frametimegpu;	// the time Of the block curve
	    varblock1[y*w*h + idy*w + idx] = (sumprod2 / prodnum[y] - powf(sumprod / prodnum[y], 2.0)) / (prodnum[y] * powf(directm * delayedm, 2.0));	// value of the block curve
	}

	for (int x = 0; x < blocknumgpu; x++) {
	    varblock1[x*w*h + idy*w + idx] = sqrt(varblock1[x*w*h + idy*w + idx]); // calculate the standard deviation
            varblock2[x*w*h + idy*w + idx] = varblock1[x*w*h + idy*w + idx] / sqrt((double)2 * (prodnum[x] - 1)); // calculate the error 
            __syncthreads();
	    upper[x*w*h + idy*w + idx] = varblock1[x*w*h + idy*w + idx] + varblock2[x*w*h + idy*w + idx]; // upper and lower quartile
	    lower[x*w*h + idy*w + idx] = varblock1[x*w*h + idy*w + idx] - varblock2[x*w*h + idy*w + idx];
	}

        // determine index where blocking criteria are fulfilled
	for (int x = 0; x < blocknumgpu - 1; x++) { // do neighboring points have overlapping error bars?
            if (upper[x*w*h + idy*w + idx] > lower[(x + 1)*w*h + idy*w + idx] && upper[(x + 1)*w*h + idy*w + idx] > lower[x*w*h + idy*w + idx]) {
                crt[x*w*h + idy*w + idx] = 1;
            }
        }

	for (int x = 0; x < blocknumgpu - 2; x++) { // do three adjacent points have overlapping error bars?
            if (crt[x*w*h + idy*w + idx] * crt[(x + 1)*w*h + idy*w + idx] == 1) {
                cr12[x*w*h + idy*w + idx] = 1;
            }
        }

	for (int x = 0; x < blocknumgpu - 1; x++) { // do neighboring points have a positive difference (increasing SD)?
            if (varblock1[(x + 1)*w*h + idy*w + idx] - varblock1[x*w*h + idy*w + idx] > 0) {
                diffpos[x*w*h + idy*w + idx] = 1;
            }
        }

        for (int x = 0; x < blocknumgpu - 2; x++) { // do three neighboring points monotonically increase?
            if (diffpos[x*w*h + idy*w + idx] * diffpos[(x + 1)*w*h + idy*w + idx] == 1) {
                cr3[x*w*h + idy*w + idx] = 1;
            }
        }

        for (int x = 0; x < blocknumgpu - 2; x++) { // find the last triple of points with monotonically increasing differences and non-overlapping error bars
            if ((cr3[x*w*h + idy*w + idx] == 1 && cr12[x*w*h + idy*w + idx] == 0)) {
                last0 = x;
            }
        }

        for (int x = 0; x <= last0; x++) { // indices of two pairs that pass criterion 1 an 2
          cr12[x*w*h + idy*w + idx] = 0;
        }

        cr12[(blocknumgpu - 3)*w*h + idy*w + idx] = 0; // criterion 3, the last two points can't be part of the blocking triple
        cr12[(blocknumgpu - 4)*w*h + idy*w + idx] = 0;

        for (int x = blocknumgpu - 5; x > 0; x--) { // index of triplet with overlapping error bars and after which no other triplet has a significant monotonic increase
            if (cr12[x*w*h + idy*w + idx] == 1) {											// or 4 increasing points
                ind = x + 1;
            }
        }

        if (ind == 0) { // if optimal blocking is not possible, use maximal blocking
            blockIndS = 0;
            if (blocknumgpu - 3 > 0) {
                ind = blocknumgpu - 3;
	    } else {
                ind = blocknumgpu - 1;
            }
        } else {
	    blockIndS = 1;
        }

        ind = (int)fmax((double)ind, (double)correlatorq - 1);
        data1[idy*w + idx] = (double)ind;
        data1[w*h + idy*w + idx] = (double)blockIndS;

    } // if ((idx < w) && (idy < h))
} // calcacf3

// initialize/set all values in the int array to value
void initializeintarr(int * array, unsigned long size, int value){
    for (unsigned long i = 0; i < size; i++){
        *(array+i) = value;
    }
}

// initialize/set all values in the double array to value
void initializedoublearr(double * array, unsigned long size, double value){
    for (unsigned long i = 0; i < size; i++){
        *(array+i) = value;
    }
}

/*
 * Calls calcacf3() and calcacf2()
 *
 * Class:     gpufitImFCS_GpufitImFCS
 * Method:    calcACF
 * Signature: ([F[D[D[D[D[D[ILgpufitImFCS/GpufitImFCS/ACFParameters;)V
 */
void JNICALL Java_gpufitImFCS_GpufitImFCS_calcACF(JNIEnv * env, jclass cls, jfloatArray pixels, jdoubleArray pixels1, jdoubleArray blockvararray, jdoubleArray NBmeanGPU, jdoubleArray NBcovarianceGPU, jdoubleArray blocked1D, jdoubleArray bleachcorr_params, jdoubleArray samp, jintArray lag, jobject ACFInputParams){
    // NOTE: outputs are stored in pixels1 and blockvararray arrays.
    // NOTE: Cpixels1 and Cblockvararray are temporary arrays to store output values before passing them to Java output arrays, by reference.

    // host arrays
    //------------------ common ---------------------------
    jfloat *Cpixels;
    double *Cpixels1;
    float *prod;
    double *Cbleachcorr_params;
    jdouble *Csamp;
    jint *Clag;
    double *Cblocked1D; // to copy Cpixels1 after calcacf3
    jboolean isNBcalculation;

    //------------------ calcacf3 ---------------------------      
    double *prodnum;
    double *blocksd;
    double *upper;
    double *lower;
    int *crt;
    int *cr12;
    int *cr3;
    int *diffpos;
    double *varblock0;
    double *varblock1;
    double *varblock2;

    //------------------ calcacf2 ---------------------------
    int *prodnumarray;
    int *indexarray;
    double *Cblockvararray;
    double *blocksdarray;
    int *pnumgpu;

    //--------------- N & B calculations in calcacf2 --------------------
    double *CNBmeanGPU;
    double *CNBcovarianceGPU;

    // CUDA arrays
    //------------------ common ---------------------------
    float *d_Cpixels;
    double *d_Cpixels1;
    double *d_Cbleachcorr_params;
    double *d_Csamp;
    int *d_Clag; 
    float *d_prod;

    //------------------ calcacf3 ---------------------------      
    double *d_prodnum;
    double *d_blocksd;
    double *d_upper;
    double *d_lower;
    int *d_crt;
    int *d_cr12;
    int *d_cr3;
    int *d_diffpos;
    double *d_varblock0;
    double *d_varblock1;
    double *d_varblock2;

    //------------------ calcacf2 ---------------------------
    int *d_prodnumarray;
    int *d_indexarray;
    double *d_Cblockvararray;
    double *d_blocksdarray;
    int *d_pnumgpu;

    //--------------- N & B calculations in calcacf2 --------------------
    double *d_NBmeanGPU;
    double *d_NBcovarianceGPU;

    try{
      int device = 0;
      CUDA_CHECK_STATUS(hipSetDevice(device));	

      //reference: https://devblogs.nvidia.com/how-overlap-data-transfers-cuda-cc/
      // NOTE: using stream to control synchronous processing of calcacf2 and calcacf3 kernels.
      // based on the reference, we can potentially use it to speed up the transfer of data process.
      hipStream_t stream;
      CUDA_CHECK_STATUS(hipStreamCreate( &stream ));

      size_t SIZEINT = sizeof(int);
      size_t SIZEFLOAT = sizeof(float);
      size_t SIZEDOUBLE = sizeof(double);

      // input arrays required for calculations.
      Cpixels = env->GetFloatArrayElements(pixels, NULL);
      Cbleachcorr_params = env->GetDoubleArrayElements(bleachcorr_params, NULL);
      Csamp = env->GetDoubleArrayElements(samp, NULL);
      Clag = env->GetIntArrayElements(lag, NULL);

//      jint lensamp = env->GetArrayLength(samp);
//      jint lenlag = env->GetArrayLength(lag);
    
      // get parameters that are required for the ACF calculations from the ACFInputParams object
      jclass ACFInputParamsCls = env->GetObjectClass(ACFInputParams);

      jfieldID widthId = env->GetFieldID(ACFInputParamsCls, "width", "I");
      jfieldID heightId = env->GetFieldID(ACFInputParamsCls, "height", "I");
      jfieldID w_tempId = env->GetFieldID(ACFInputParamsCls, "w_temp", "I");
      jfieldID h_tempId = env->GetFieldID(ACFInputParamsCls, "h_temp", "I");
      jfieldID pixbinXId = env->GetFieldID(ACFInputParamsCls, "pixbinX", "I");
      jfieldID pixbinYId = env->GetFieldID(ACFInputParamsCls, "pixbinY", "I");
      jfieldID firstframeId = env->GetFieldID(ACFInputParamsCls, "firstframe", "I");
      jfieldID lastframeId = env->GetFieldID(ACFInputParamsCls, "lastframe", "I");
      jfieldID cfXDistanceId = env->GetFieldID(ACFInputParamsCls, "cfXDistance", "I");
      jfieldID cfYDistanceId = env->GetFieldID(ACFInputParamsCls, "cfYDistance", "I");
      jfieldID correlatorpId = env->GetFieldID(ACFInputParamsCls, "correlatorp", "D");
      jfieldID correlatorqId = env->GetFieldID(ACFInputParamsCls, "correlatorq", "D");
      jfieldID frametimeId = env->GetFieldID(ACFInputParamsCls, "frametime", "D");
      jfieldID backgroundId = env->GetFieldID(ACFInputParamsCls, "background", "I");
      jfieldID mtab1Id = env->GetFieldID(ACFInputParamsCls, "mtab1", "D");
      jfieldID mtabchanumminus1Id = env->GetFieldID(ACFInputParamsCls, "mtabchanumminus1", "D");
      jfieldID sampchanumminus1Id = env->GetFieldID(ACFInputParamsCls, "sampchanumminus1", "D");
      jfieldID chanumId = env->GetFieldID(ACFInputParamsCls, "chanum", "I");
      jfieldID isNBcalculationId = env->GetFieldID(ACFInputParamsCls, "isNBcalculation", "Z");
      jfieldID bleachcorr_gpuId = env->GetFieldID(ACFInputParamsCls, "bleachcorr_gpu", "Z");
      jfieldID bleachcorr_orderId = env->GetFieldID(ACFInputParamsCls, "bleachcorr_order", "I");

      jint width = env->GetIntField(ACFInputParams, widthId);
      jint height = env->GetIntField(ACFInputParams, heightId);
      jint w_temp = env->GetIntField(ACFInputParams, w_tempId);
      jint h_temp = env->GetIntField(ACFInputParams, h_tempId);
      jint pixbinX = env->GetIntField(ACFInputParams, pixbinXId);
      jint pixbinY = env->GetIntField(ACFInputParams, pixbinYId);
      jint firstframe = env->GetIntField(ACFInputParams, firstframeId);
      jint lastframe = env->GetIntField(ACFInputParams, lastframeId);
      jint cfXDistance = env->GetIntField(ACFInputParams, cfXDistanceId);
      jint cfYDistance = env->GetIntField(ACFInputParams, cfYDistanceId);
      jdouble correlatorpdbl = env->GetDoubleField(ACFInputParams, correlatorpId);
      jdouble correlatorqdbl = env->GetDoubleField(ACFInputParams, correlatorqId);
      jdouble frametime = env->GetDoubleField(ACFInputParams, frametimeId);
      jint background = env->GetIntField(ACFInputParams, backgroundId);
      jdouble mtab1 = env->GetDoubleField(ACFInputParams, mtab1Id); // mtab[1], used to calculate blocknumgpu.
      jdouble mtabchanumminus1 = env->GetDoubleField(ACFInputParams, mtabchanumminus1Id); // mtab[chanum-1], used to calculate pnumgpu[counter_indexarray]
      jdouble sampchanumminus1 = env->GetDoubleField(ACFInputParams, sampchanumminus1Id); // samp[chanum-1], used to calculate pnumgpu[counter_indexarray]
      jint chanum = env->GetIntField(ACFInputParams, chanumId);
      isNBcalculation = env->GetBooleanField(ACFInputParams, isNBcalculationId);      
      jboolean bleachcorr_gpu = env->GetBooleanField(ACFInputParams, bleachcorr_gpuId);      
      jint bleachcorr_order = env->GetIntField(ACFInputParams, bleachcorr_orderId);

      // initialize parameters
      int correlatorp = (int) correlatorpdbl;
      int correlatorq = (int) correlatorqdbl;
      int framediff = lastframe - firstframe + 1;
      size_t size = w_temp * h_temp * framediff * SIZEFLOAT;
      size_t size1 = width * height * chanum * SIZEDOUBLE;

      int blocklaggpu = 1;

      size_t size2 = framediff * width * height * SIZEFLOAT;
      size_t sizeblockvararray = chanum * width * height * SIZEDOUBLE;
          
      int blocknumgpu = (int) (floor(log(mtab1)/log(2)) - 2);

      // blockSize and gridSize
      int BLKSIZEXY = 16;
      int a = ( width > height ) ? width : height;
      int GRIDSIZEXY = (a + BLKSIZEXY -1) / BLKSIZEXY;

      dim3 blockSize(BLKSIZEXY, BLKSIZEXY, 1);
      dim3 gridSize(GRIDSIZEXY, GRIDSIZEXY, 1);

      int b = ( w_temp > h_temp ) ? w_temp : h_temp;
      int GRIDSIZEXY_Input = (b + BLKSIZEXY -1) / BLKSIZEXY;
      
      dim3 gridSize_Input(GRIDSIZEXY_Input, GRIDSIZEXY_Input, 1);

      // dynamic memory allocation and/or initialization
      //------------------ common parameters ---------------------------
      Cpixels1 = (double *)malloc(size1);
      prod = (float *)malloc(size2);
      Cblocked1D = (double *)malloc(size1); // Cblocked1D copies Cpixels1 array after calcacf3 calculation and not required GPU memory.

      //------------------ calcacf3 ---------------------------
      // Using if (!isNBcalculation) to comment out this section will cause warnings "may be used uninitialized in this function" being shown during compilation
      // instead, we will still initialize the arrays with malloc, but skip the initializedoublearr, initializeintarr functions.
      prodnum = (double *)malloc(blocknumgpu * SIZEDOUBLE);
      blocksd = (double *)malloc(sizeblockvararray);
      upper = (double *)malloc(blocknumgpu*width*height*SIZEDOUBLE);
      lower = (double *)malloc(blocknumgpu*width*height*SIZEDOUBLE);
      crt = (int *)malloc((blocknumgpu-1)*width*height*SIZEINT);
      cr12 = (int *)malloc((blocknumgpu-2)*width*height*SIZEINT);
      cr3 = (int *)malloc((blocknumgpu-2)*width*height*SIZEINT);
      diffpos = (int *)malloc((blocknumgpu-1)*width*height*SIZEINT);
      varblock0 = (double *)malloc(blocknumgpu*width*height*SIZEDOUBLE);
      varblock1 = (double *)malloc(blocknumgpu*width*height*SIZEDOUBLE);
      varblock2 = (double *)malloc(blocknumgpu*width*height*SIZEDOUBLE);

      if (!isNBcalculation) { 
          initializedoublearr(prodnum, blocknumgpu, 1.0); // initialize all values in prodnum to 1.0
          initializedoublearr(blocksd, chanum * width * height, 0.0);
          initializedoublearr(upper, blocknumgpu*width*height, 0.0);
          initializedoublearr(lower, blocknumgpu*width*height, 0.0);
          initializeintarr(crt, (blocknumgpu-1)*width*height, 0);
          initializeintarr(cr12, (blocknumgpu-2)*width*height, 0);
          initializeintarr(cr3, (blocknumgpu-2)*width*height, 0);
          initializeintarr(diffpos, (blocknumgpu-1)*width*height, 0);
          initializedoublearr(varblock0, blocknumgpu*width*height, 0.0);   
          initializedoublearr(varblock1, blocknumgpu*width*height, 0.0);   
          initializedoublearr(varblock2, blocknumgpu*width*height, 0.0);   
      }

      //------------------ calcacf2 ---------------------------
      prodnumarray = (int *)malloc(chanum*width*height*SIZEINT);
      indexarray = (int *)malloc(width*height*SIZEINT);
      Cblockvararray = (double *)malloc(sizeblockvararray);
      blocksdarray = (double *)malloc(sizeblockvararray);
      pnumgpu = (int *)malloc(width*height*SIZEINT);

      //--------------- N & B calculations in calcacf2 --------------------
      CNBmeanGPU = (double *)malloc( width * height * SIZEDOUBLE );
      CNBcovarianceGPU = (double *)malloc( width * height * SIZEDOUBLE );

      if (isNBcalculation) { 
          initializedoublearr(CNBmeanGPU, width * height, 0.0);
          initializedoublearr(CNBcovarianceGPU, width * height, 0.0);
      }

      // ------------------- perform calcacf3 calculation -------------------
      // Allocate memory on GPU for common arrays
      hipMalloc((void **)&d_Cpixels, size);
      hipMalloc((void **)&d_Cpixels1, size1);
      hipMalloc((void **)&d_Cbleachcorr_params, w_temp * h_temp * bleachcorr_order * SIZEDOUBLE);
      hipMalloc((void **)&d_Csamp, chanum * SIZEDOUBLE);
      hipMalloc((void **)&d_Clag, chanum * SIZEINT);
      hipMalloc((void **)&d_prod, size2);

      // Allocate memory on GPU for calcacf3
      if (!isNBcalculation) {
          hipMalloc((void **)&d_prodnum, blocknumgpu * SIZEDOUBLE);
          hipMalloc((void **)&d_blocksd, sizeblockvararray);
          hipMalloc((void **)&d_upper, blocknumgpu*width*height*SIZEDOUBLE);
          hipMalloc((void **)&d_lower, blocknumgpu*width*height*SIZEDOUBLE);
          hipMalloc((void **)&d_crt, (blocknumgpu-1)*width*height*SIZEINT);
          hipMalloc((void **)&d_cr12, (blocknumgpu-2)*width*height*SIZEINT);
          hipMalloc((void **)&d_cr3, (blocknumgpu-2)*width*height*SIZEINT);
          hipMalloc((void **)&d_diffpos, (blocknumgpu-1)*width*height*SIZEINT);
          hipMalloc((void **)&d_varblock0, blocknumgpu*width*height*SIZEDOUBLE);
          hipMalloc((void **)&d_varblock1, blocknumgpu*width*height*SIZEDOUBLE);
          hipMalloc((void **)&d_varblock2, blocknumgpu*width*height*SIZEDOUBLE);
      }

      // copy to GPU for common arrays
      CUDA_CHECK_STATUS(hipMemcpy(d_Cpixels, Cpixels, size, hipMemcpyHostToDevice));
      CUDA_CHECK_STATUS(hipMemcpy(d_Cpixels1, Cpixels1, size1, hipMemcpyHostToDevice)); 
      CUDA_CHECK_STATUS(hipMemcpy(d_Cbleachcorr_params, Cbleachcorr_params, w_temp * h_temp * bleachcorr_order * SIZEDOUBLE, hipMemcpyHostToDevice));
      CUDA_CHECK_STATUS(hipMemcpy(d_Csamp, Csamp, chanum * SIZEDOUBLE, hipMemcpyHostToDevice)); 
      CUDA_CHECK_STATUS(hipMemcpy(d_Clag, Clag, chanum * SIZEINT, hipMemcpyHostToDevice)); 
      // NOTE: When pixels + prod are larger than half the memory available on CPU, this hipMemcpy function will fail. The half memory limit is restriction in Java? If we comment this line, we will encounter a different error when the kernel calcacf3 is run.
      CUDA_CHECK_STATUS(hipMemcpy(d_prod, prod, size2, hipMemcpyHostToDevice)); 

      // copy to GPU for calcacf3      
      if (!isNBcalculation) {
          CUDA_CHECK_STATUS(hipMemcpy(d_prodnum, prodnum, blocknumgpu*SIZEDOUBLE, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_blocksd, blocksd, sizeblockvararray, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_upper, upper, blocknumgpu*width*height*SIZEDOUBLE, hipMemcpyHostToDevice));
          CUDA_CHECK_STATUS(hipMemcpy(d_lower, lower, blocknumgpu*width*height*SIZEDOUBLE, hipMemcpyHostToDevice));  
          CUDA_CHECK_STATUS(hipMemcpy(d_crt, crt, (blocknumgpu-1)*width*height*SIZEINT, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_cr12, cr12, (blocknumgpu-2)*width*height*SIZEINT, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_cr3, cr3, (blocknumgpu-2)*width*height*SIZEINT, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_diffpos, diffpos, (blocknumgpu-1)*width*height*SIZEINT, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_varblock0, varblock0, blocknumgpu*width*height*SIZEDOUBLE, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_varblock1, varblock1, blocknumgpu*width*height*SIZEDOUBLE, hipMemcpyHostToDevice)); 
          CUDA_CHECK_STATUS(hipMemcpy(d_varblock2, varblock2, blocknumgpu*width*height*SIZEDOUBLE, hipMemcpyHostToDevice)); 
      }

      //running kernel for calcacf3
      if (!isNBcalculation) {
          if(bleachcorr_gpu) {
              bleachcorrection<<<gridSize_Input, blockSize, 0, stream>>>(d_Cpixels, w_temp, h_temp, framediff, bleachcorr_order, frametime, d_Cbleachcorr_params);
              hipDeviceSynchronize();
              CUDA_CHECK_STATUS(hipGetLastError());
          }

          calcacf3<<<gridSize, blockSize, 0, stream>>>(d_Cpixels, cfXDistance, cfYDistance, blocklaggpu, width, height, w_temp, h_temp, pixbinX, pixbinY, framediff, correlatorp, correlatorq, chanum, frametime, d_Cpixels1, d_prod, d_prodnum, d_blocksd, d_upper, d_lower, d_crt, d_cr12, d_cr3, d_diffpos, d_varblock0, d_varblock1, d_varblock2, d_Csamp, d_Clag);

          hipDeviceSynchronize();
          CUDA_CHECK_STATUS(hipGetLastError());

          // copy memory from device to host for calcacf3
          CUDA_CHECK_STATUS(hipMemcpy(Cpixels1, d_Cpixels1, size1, hipMemcpyDeviceToHost));

          // CUDA release memory for calcacf3
          hipFree(d_prodnum); hipFree(d_blocksd); hipFree(d_upper); hipFree(d_lower); hipFree(d_crt); hipFree(d_cr12); hipFree(d_cr3); hipFree(d_diffpos); 
          hipFree(d_varblock0); hipFree(d_varblock1); hipFree(d_varblock2); 

          // copy results in Cpixels1 to Cblocked1D. Values in Cpixels1 will be overwritten in calcacf2 calculation
          // memcpy(Cblocked1D, Cpixels1, size1);
          for (int i = 0; i < width * height * chanum; i++){
            *(Cblocked1D+i) = *(Cpixels1+i);
          }

          // initialize the values in indexarray and pnumgpu
          int counter_indexarray = 0;
          for (int y = 0; y < height; y++){
                for (int x = 0; x < width; x++){
                    *(indexarray + counter_indexarray) =  (int) *(Cpixels1 + (y*width + x));
          
                    //minimum number of products given the used correlator structure and the blockIndex ind
                    double tempval = *(indexarray + counter_indexarray) - log(sampchanumminus1) / log(2);
                    if (tempval < 0){
                        tempval = 0;
                    }
                    *(pnumgpu + counter_indexarray) = (int) floor(mtabchanumminus1/pow(2,tempval));
                    counter_indexarray = counter_indexarray + 1;
                }
          }

      } // if (!isNBcalculation)

      // ------------------- perform calcacf2 calculation -------------------

      // The pixel values in d_Cpixels has changed in calcacf3, reallocate original Cpixels to d_Cpixels
      CUDA_CHECK_STATUS(hipMemcpy(d_Cpixels, Cpixels, size, hipMemcpyHostToDevice));

      // Allocate memory on GPU for calcacf2
      hipMalloc((void **)&d_prodnumarray, chanum*width*height*SIZEINT);
      hipMalloc((void **)&d_indexarray, width*height*SIZEINT);
      hipMalloc((void **)&d_Cblockvararray, sizeblockvararray);
      hipMalloc((void **)&d_blocksdarray, sizeblockvararray);
      hipMalloc((void **)&d_pnumgpu, width*height*SIZEINT);

      // Allocate memory on GPU for N & B calculations in calcacf2
      if (isNBcalculation) {
          hipMalloc((void **)&d_NBmeanGPU, width * height * SIZEDOUBLE);
          hipMalloc((void **)&d_NBcovarianceGPU, width * height * SIZEDOUBLE);
      }

      // copy to GPU for calcacf2
      CUDA_CHECK_STATUS(hipMemcpy(d_indexarray, indexarray, width*height*SIZEINT, hipMemcpyHostToDevice)); 
      CUDA_CHECK_STATUS(hipMemcpy(d_prodnumarray, prodnumarray, chanum*width*height*SIZEINT, hipMemcpyHostToDevice)); 
      CUDA_CHECK_STATUS(hipMemcpy(d_Cblockvararray, Cblockvararray, sizeblockvararray, hipMemcpyHostToDevice)); 
      CUDA_CHECK_STATUS(hipMemcpy(d_blocksdarray, blocksdarray, sizeblockvararray, hipMemcpyHostToDevice)); 
      CUDA_CHECK_STATUS(hipMemcpy(d_pnumgpu, pnumgpu, width*height*SIZEINT, hipMemcpyHostToDevice)); 

      // copy to GPU for N & B calculatiosn in calcacf2
      if (isNBcalculation) {
          CUDA_CHECK_STATUS(hipMemcpy(d_NBmeanGPU, CNBmeanGPU, width * height * SIZEDOUBLE, hipMemcpyHostToDevice));
          CUDA_CHECK_STATUS(hipMemcpy(d_NBcovarianceGPU, CNBcovarianceGPU, width * height * SIZEDOUBLE, hipMemcpyHostToDevice));
      }

      //running kernel for calcacf2
      if(bleachcorr_gpu) {
          bleachcorrection<<<gridSize_Input, blockSize, 0, stream>>>(d_Cpixels, w_temp, h_temp, framediff, bleachcorr_order, frametime, d_Cbleachcorr_params);
          hipDeviceSynchronize();
          CUDA_CHECK_STATUS(hipGetLastError());
      }

      int numbin = framediff;		// number Of data points When they are binned
      int currentIncrement = 1;
      int ctbin = 0;
      bool runthis;

      int calcacf2_x_start = (isNBcalculation) ? 1 : 0;
      int calcacf2_x_end = (isNBcalculation) ? 2 : chanum;

      for (int x = calcacf2_x_start; x < calcacf2_x_end; x++) {
          runthis = false;
	  if (currentIncrement != Csamp[x]){ // check whether the channel width has changed
              // Set the currentIncrement accordingly
              numbin = (int)floor((double)numbin / 2.0);
              currentIncrement = (int)Csamp[x];
              ctbin++;
              runthis = true;
          }

          if (runthis){ // check whether the channel width has changed
              calcacf2a<<<gridSize_Input, blockSize, 0, stream>>>(d_Cpixels, w_temp, h_temp, numbin);
              hipDeviceSynchronize();
              CUDA_CHECK_STATUS(hipGetLastError());
          }

          calcacf2b<<<gridSize, blockSize, 0, stream>>>(d_Cpixels, cfXDistance, cfYDistance, width, height, w_temp, h_temp, pixbinX, pixbinY, d_Cpixels1, d_prod, d_Clag, d_prodnumarray, d_indexarray, d_Cblockvararray, d_blocksdarray, d_pnumgpu, x, numbin, currentIncrement, ctbin, isNBcalculation, d_NBmeanGPU, d_NBcovarianceGPU);
          hipDeviceSynchronize();
          CUDA_CHECK_STATUS(hipGetLastError());
      }

      // copy memory from device to host for calcacf2
      if (isNBcalculation) { 
          CUDA_CHECK_STATUS(hipMemcpy(CNBmeanGPU, d_NBmeanGPU, width * height * SIZEDOUBLE, hipMemcpyDeviceToHost));
          CUDA_CHECK_STATUS(hipMemcpy(CNBcovarianceGPU, d_NBcovarianceGPU, width * height * SIZEDOUBLE, hipMemcpyDeviceToHost));
      } else {
          CUDA_CHECK_STATUS(hipMemcpy(Cpixels1, d_Cpixels1, size1, hipMemcpyDeviceToHost));
          CUDA_CHECK_STATUS(hipMemcpy(Cblockvararray, d_Cblockvararray, sizeblockvararray, hipMemcpyDeviceToHost));
      }

      // CUDA release memory for calcacf2
      hipFree(d_prodnumarray); hipFree(d_indexarray); hipFree(d_Cblockvararray); hipFree(d_blocksdarray); hipFree(d_pnumgpu);

      // CUDA release memory for N & B calculations in calcacf2
      if (isNBcalculation) { hipFree(d_NBmeanGPU); hipFree(d_NBcovarianceGPU); }

      // CUDA release memory for all
      hipFree(d_Cpixels); hipFree(d_Cpixels1); hipFree(d_Cbleachcorr_params); hipFree(d_Csamp); hipFree(d_Clag); hipFree(d_prod);

      CUDA_CHECK_STATUS(hipStreamDestroy( stream ));

      // Reference: https://github.com/zchee/cuda-sample/blob/master/0_Simple/simpleMultiGPU/simpleMultiGPU.cu 
      // hipDeviceReset causes the driver to clean up all state. While
      // not mandatory in normal operation, it is good practice.  It is also
      // needed to ensure correct operation when the application is being
      // profiled. Calling hipDeviceReset causes all profile data to be
      // flushed before the application exits
      hipDeviceReset(); 

      // copy values to Java output arrays.
      env->SetDoubleArrayRegion(pixels1, 0 , width * height * chanum, Cpixels1);  
      env->SetDoubleArrayRegion(blockvararray, 0 , chanum * width * height, Cblockvararray);
      env->SetDoubleArrayRegion(blocked1D, 0 , width * height * chanum, Cblocked1D);  

      if (isNBcalculation) {
          env->SetDoubleArrayRegion(NBmeanGPU, 0 , width * height , CNBmeanGPU); 
          env->SetDoubleArrayRegion(NBcovarianceGPU, 0 , width * height , CNBcovarianceGPU); 
      }

      // free all pointers
      free(prod); free(prodnum); free(blocksd); free(upper); free(lower); free(crt); free(cr12); free(cr3); free(diffpos); free(varblock0); free(varblock1); free(varblock2); 
      free(indexarray); free(prodnumarray); free(Cblockvararray); free(blocksdarray); free(pnumgpu);
      free(Cpixels1); free(Cblocked1D);
      free(CNBmeanGPU); free(CNBcovarianceGPU);
      
      // release resources
      env->ReleaseFloatArrayElements(pixels, Cpixels, 0);
      env->ReleaseDoubleArrayElements(bleachcorr_params, Cbleachcorr_params, 0);
      env->ReleaseDoubleArrayElements(samp, Csamp, 0);
      env->ReleaseIntArrayElements(lag, Clag, 0);

      return;  

    } catch (std::runtime_error & e)  {
      // see: https://www.rgagnon.com/javadetails/java-0323.html
      jclass Exception = env->FindClass("java/lang/Exception");
      env->ThrowNew(Exception, e.what());

      // CUDA release memory for calcacf3
      if (!isNBcalculation) {
          hipFree(d_prodnum); hipFree(d_blocksd); hipFree(d_upper); hipFree(d_lower); hipFree(d_crt); hipFree(d_cr12); hipFree(d_cr3); hipFree(d_diffpos); 
          hipFree(d_varblock0); hipFree(d_varblock1); hipFree(d_varblock2); 
      }

      // CUDA release memory for calcacf2
      hipFree(d_prodnumarray); hipFree(d_indexarray); hipFree(d_Cblockvararray); hipFree(d_blocksdarray); hipFree(d_pnumgpu);

      // CUDA release memory for N & B calculations in calcacf2
      if (isNBcalculation) { hipFree(d_NBmeanGPU); hipFree(d_NBcovarianceGPU); }

      // CUDA release memory for all
      hipFree(d_Cpixels); hipFree(d_Cpixels1); hipFree(d_Cbleachcorr_params); hipFree(d_Csamp); hipFree(d_Clag); hipFree(d_prod); 

      hipDeviceReset(); 

      // free all pointers
      free(prod); free(prodnum); free(blocksd); free(upper); free(lower); free(crt); free(cr12); free(cr3); free(diffpos); free(varblock0); free(varblock1); free(varblock2); 
      free(indexarray); free(prodnumarray); free(Cblockvararray); free(blocksdarray); free(pnumgpu);
      free(Cpixels1); free(Cblocked1D);
      free(CNBmeanGPU); free(CNBcovarianceGPU);

      // release resources
      env->ReleaseFloatArrayElements(pixels, Cpixels, 0);
      env->ReleaseDoubleArrayElements(bleachcorr_params, Cbleachcorr_params, 0);
      env->ReleaseDoubleArrayElements(samp, Csamp, 0);
      env->ReleaseIntArrayElements(lag, Clag, 0);

      return;  
    }
}

/* ------------------------------------------
AUTOCORRELATION SINGLE DIMENSION ARRAY CALCULATION END
------------------------------------------ */

/* -------------------------------------------------------------------------------------------------------
* from com_github_gpufit_Gpufit.cpp END
------------------------------------------------------------------------------------------------------- */
